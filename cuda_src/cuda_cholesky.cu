#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

double *M;

#define cudacall(call) \
	{ \
	hipError_t err = (call);                                                                                               \
	if(hipSuccess != err)                                                                                                  \
	{                                                                                                                       \
		fprintf(stderr,"CUDA Error:\nFile = %s\nLine = %d\nReason = %s\n", __FILE__, __LINE__, hipGetErrorString(err));    \
		hipDeviceReset();                                                                                                  \
		exit(EXIT_FAILURE);                                                                                                 \
	}                                                                                                                       \
	} \


	__global__ void
hardcoded_cholesky_2x2(double *M, int dim, int start_id)
{
	int i_index = (blockIdx.x * blockDim.x + threadIdx.x);
	int j_index = (blockIdx.y * blockDim.y + threadIdx.y);

	printf("In kernel\n");
	printf("dim is %d, i_index is %d, j_index is %d\n", dim, i_index, j_index);

	for (int i = 0; i < dim; i++)
	{
		for (int j = 0; j < dim; j++)
		{
			printf("%lf ", M[i * dim + j]);
		}
		printf("\n");
	}
}


void get_symmetric_matrix_1d(double *M, double **matrix1, double **matrix2, int dim) {

	srand(time(NULL));
	for (int i = 0; i < dim; i++){
		for (int j = 0; j < dim; j++){
			matrix1[i][j] = rand() % 100 + 1;
			matrix2[j][i] = matrix1[i][j];
		}
	}

	for (int i = 0; i < dim; i++){
		for(int j = 0; j < dim; j++){
			for(int k = 0; k < dim; k++){
				M[i * dim + j] += matrix1[i][k]*matrix2[k][j];
			}
		}
	}
}

void setup(int dim)
{
	int deviceCount = 0;
	bool isFastGPU = false;
	std::string name;
	hipError_t err = hipGetDeviceCount(&deviceCount);

	printf("---------------------------------------------------------\n");
	printf("Initializing CUDA for cuGP\n");
	printf("Found %d CUDA devices\n", deviceCount);

	for (int i=0; i<deviceCount; i++) {
		hipDeviceProp_t deviceProps;
		hipGetDeviceProperties(&deviceProps, i);
		name = deviceProps.name;
		if (name.compare("GeForce GTX 480") == 0
				|| name.compare("GeForce GTX 670") == 0
				|| name.compare("GeForce GTX 780") == 0)
		{
			isFastGPU = true;
		}

		printf("Device %d: %s\n", i, deviceProps.name);
		printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
		printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
		printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
	}
	printf("---------------------------------------------------------\n");
	if (!isFastGPU)
	{
		printf("WARNING: "
				"You're not running on a fast GPU, please consider using "
				"NVIDIA GTX 480, 670 or 780.\n");
		printf("---------------------------------------------------------\n");
	}

	// By this time the scene should be loaded.  Now copy all the key
	// data structures into device memory so they are accessible to
	// CUDA kernels
	//
	// See the CUDA Programmer's Guide for descriptions of
	// hipMalloc and hipMemcpy

	double *temp_m, **m1, **m2;

	temp_m = new double[dim * dim];

	m1 = new double *[dim];
	m2 = new double *[dim];

	for (int i = 0; i < dim; i++)
	{
		m1[i] = new double[dim];
		m2[i] = new double[dim];
	}

	get_symmetric_matrix_1d(temp_m, m1, m2, dim);

	printf("Generated matrix in host is \n");
	for (int i = 0; i < dim; i++)
	{
		for (int j = 0; j < dim; j++)
		{
			printf("%lf ", temp_m[i * dim + j]);
		}
		printf("\n");
	}

	cudacall(hipMalloc(&M, sizeof(double) * dim * dim));

	cudacall(hipMemcpy(M, temp_m, sizeof(double) * dim * dim, hipMemcpyHostToDevice));

	// Initialize parameters in constant memory.  We didn't talk about
	// constant memory in class, but the use of read-only constant
	// memory here is an optimization over just sticking these values
	// in device global memory.  NVIDIA GPUs have a few special tricks
	// for optimizing access to constant memory.  Using global memory
	// here would have worked just as well.  See the Programmer's
	// Guide for more information about constant memory.

	/*GlobalConstants params;
	  params.sceneName = sceneName;
	  params.numCircles = numCircles;
	  params.imageWidth = image->width;
	  params.imageHeight = image->height;
	  params.position = cudaDevicePosition;
	  params.velocity = cudaDeviceVelocity;
	  params.color = cudaDeviceColor;
	  params.radius = cudaDeviceRadius;
	  params.imageData = cudaDeviceImageData;

	  hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));

	// also need to copy over the noise lookup tables, so we can
	// implement noise on the GPU
	int* permX;
	int* permY;
	float* value1D;
	getNoiseTables(&permX, &permY, &value1D);
	hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseXPermutationTable), permX, sizeof(int) * 256);
	hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseYPermutationTable), permY, sizeof(int) * 256);
	hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoise1DValueTable), value1D, sizeof(float) * 256);

	// last, copy over the color table that's used by the shading
	// function for circles in the snowflake demo

	float lookupTable[COLOR_MAP_SIZE][3] = {
	{1.f, 1.f, 1.f},
	{1.f, 1.f, 1.f},
	{.8f, .9f, 1.f},
	{.8f, .9f, 1.f},
	{.8f, 0.8f, 1.f},
	};

	hipMemcpyToSymbol(HIP_SYMBOL(cuConstColorRamp), lookupTable, sizeof(float) * 3 * COLOR_MAP_SIZE);

	if(hipMalloc(&table, sizeof(int) * numCircles * image->width * image->height / 16 / 16) != hipSuccess )
	printf("The giant malloc failed.\n");

	if (hipMalloc(&counters, sizeof(int) * image->height * image->width / 16 / 16) != hipSuccess)
	printf("The counter malloc fialed.\n");

	if (hipMalloc(&flagarray, sizeof(int) * numCircles) != hipSuccess)
	printf("The flag array malloc failed.\n");

	hipMemset((void *)flagarray, 0, sizeof(int) * numCircles); */
}


void run_kernel()
{
	int dim = 8, start_id;
	setup(dim);
	printf("Call kernel\n");
	// hardcoded_cholesky_2x2<<<1,1>>>();
	
	start_id = 0;

	hardcoded_cholesky_2x2<<<1, 1>>>(M, dim, start_id);
	hipDeviceSynchronize();
	printf("Kernel call done\n");
}
