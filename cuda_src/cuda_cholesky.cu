#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <cstdlib>
#include "../common/cycleTimer.h"
#include <fstream>


#define INPUT_FILE "../cpp_serial_gp/input_10.txt"
#define filename "sym5000.txt"

double *temp_m; 
double *orig_sym; //orig matrix for reference
 
double *mt;
double *mt_transpose;

double *M;
double *a11;
double *a21_transpose;
double *l21_transpose_from_fs;
double *l21;
double *l22_temp;  //This is for updating a22

#define cudacall(call) \
{ \
	hipError_t err = (call);                                                                                               \
	if(hipSuccess != err)                                                                                                  \
	{                                                                                                                       \
		fprintf(stderr,"CUDA Error:\nFile = %s\nLine = %d\nReason = %s\n", __FILE__, __LINE__, hipGetErrorString(err));    \
		hipDeviceReset();                                                                                                  \
		exit(EXIT_FAILURE);                                                                                                 \
	}                                                                                                                       \
} \


__global__ void check_forward_sub_vector(double *L, double *x, double *b, int N){

	int i_index = (blockIdx.x * blockDim.x + threadIdx.x);
	int j_index = (blockIdx.y * blockDim.y + threadIdx.y);
	if (i_index >= 1)
		return;
	
	for(int i = 0; i < N; i++) {
                double temp = 0.0;
                for(int j = 0; j < N; j++) {
                        temp += L[i*N + j] * x[j];
                }
                printf("%lf - %lf = %lf\n", temp, b[i], temp - b[i]);
        }
}
__global__ void check_backward_sub_vector(double *L, double *y, double *b, int N){

	int i_index = (blockIdx.x * blockDim.x + threadIdx.x);
	int j_index = (blockIdx.y * blockDim.y + threadIdx.y);
	if (i_index >= 1)
		return;
	
	for(int i = 0; i < N; i++) {
                double temp = 0.0;
                for(int j = 0; j < N; j++) {
                        temp += L[j * N + i] * y[j];
                }
                printf("%lf - %lf = %lf\n", temp, b[i], temp - b[i]);
        }
}


// We want to solve for "output", such that 
//	 lowert_mat * output = b;
__global__ void forward_substitution_vector(double *lowert_mat, double *b, double *output, int N){

	int i_index = (blockIdx.x * blockDim.x + threadIdx.x);
	int j_index = (blockIdx.y * blockDim.y + threadIdx.y);
	if (i_index >= 1)
		return;
	
	/*
	// Forward solve L * temp = y
        for (int i = 0; i < n; i++){
                temp[i] = y[i];
                for (int j = 0; j < i; j++)
                {
                        temp[i] -= L[i][j] * temp[j];
                }
                temp[i] /= L[i][i];
        }
	*/
	for(int i = 0 ; i < N ; i++){
		output[i] = b[i];	
		for (int j = 0 ; j < i; j++){
			output[i] -= lowert_mat[i * N + j] * output[j];
		}
		output[i] /= lowert_mat[i*N + i];
	}
}


// We want lower_mat.transpose() * output = b;
// NOTE: WE ARE NOT PASSING AN UPPER TRIANGULAR MATRIX (which would have been the case in a general implementation)
__global__ void backward_substitution_vector(double *lowert_mat, double *b, double *output, int N){

	int i_index = (blockIdx.x * blockDim.x + threadIdx.x);
	int j_index = (blockIdx.y * blockDim.y + threadIdx.y);
	if (i_index >= 1)
		return;
	
	/*
	// backward solve arr2 * y2 = b
	for(int i = DIM - 1; i >= 0 ; i--) {
                y2[i] = b[i];
                for(int j = i + 1; j < DIM; j++) {
                        y2[i] = y2[i] - arr2[i][j] * y2[j];
                }
                y2[i] = y2[i] / arr2[i][i];
        }
	*/

	for(int i = N - 1; i >= 0 ; i--) {
                output[i] = b[i];
                for(int j = i + 1; j < N; j++) {
                        output[i] = output[i] - lowert_mat[j * N + i] * output[j];
                }
                output[i] /= lowert_mat[i*N + i];
        }
}
__global__ void set_upper_zero(double *M, int dim){
	
	int i_index = (blockIdx.x * blockDim.x + threadIdx.x);
	int j_index = (blockIdx.y * blockDim.y + threadIdx.y);

	if (i_index >= (dim * dim))
		return;
	
	int rowN = i_index / dim;
	int colN = i_index % dim;
		
	if(rowN >= colN) return;
	
	M[rowN * dim + colN] = 0.0;
}

	__global__ void
hardcoded_cholesky_1x1(double *M, double *a11, int dim, int b, int start_id)
{
	// TODO
	/* M[idx][idx] = sqrt(M[idx][idx]);
	a11[0][0] = M[idx][idx]; */
}

	__global__ void
hardcoded_cholesky_2x2(double *M, double *a11, int dim, int b, int start_id)
{
	int i_index = (blockIdx.x * blockDim.x + threadIdx.x);
	int j_index = (blockIdx.y * blockDim.y + threadIdx.y);

	/*
	printf("In kernel\n");
	printf("dim is %d, i_index is %d, j_index is %d, b is %d, start_id is %d\n", dim, i_index, j_index, b, start_id);

	printf("Now the 2x2 matrix is:\n");
	for (int i = 0; i < b; i++)
	{
		for (int j = 0; j < b; j++)
		{
			printf("%lf ", M[(i + start_id) * dim + j + start_id]);
		}
		printf("\n");
	}
	*/

	a11[0] = M[start_id * dim + start_id] = sqrt(M[start_id * dim + start_id]);
	a11[1] = M[start_id * dim + start_id + 1] = 0.0;
	a11[2] = M[(start_id + 1) * dim + start_id] = M[(start_id + 1) * dim + start_id] / M[start_id * dim + start_id];
	a11[3] = M[(start_id + 1) * dim + start_id + 1] = sqrt(M[(start_id + 1) * dim + start_id + 1] - M[(start_id + 1) * dim + start_id] * M[(start_id + 1) * dim + start_id]);

	//printf("printing a11 matrix\n");
	//printf("%lf %lf %lf %lf\n", a11[0], a11[1], a11[2], a11[3]);
}

__global__ void
print_matrix_kernel(double *arr, int dim1, int dim2)
{
	printf("Printing matrix:\n");
	for (int i = 0; i < dim1; i++)
	{
		for (int j = 0; j < dim2; j++)
		{
			printf("%lf ", arr[i * dim2 + j]);
		}
		printf("\n");
	}
}

__global__ void
take_a21_transpose(double *M, double *a21_transpose, int dim, int b, int start_id) {
	int i_index = (blockIdx.x * blockDim.x + threadIdx.x);
	int j_index = (blockIdx.y * blockDim.y + threadIdx.y);

	if (i_index >= (dim - b - start_id) * b)
		return;

	//printf("In a21_transpose, i_index is %d, j_index is %d\n", i_index, j_index);

	int input_row, input_col, target_row, target_col, row_offset_by_thread, col_offset_by_thread;

	// TODO replace i_index by a generic construct because it may involve blocks and grids
	row_offset_by_thread = i_index / b;
	input_row = start_id + b + row_offset_by_thread;

	col_offset_by_thread = i_index % b;
	input_col = start_id + col_offset_by_thread;

	target_row = i_index % b;
	target_col = i_index / b;

	a21_transpose[target_row * (dim - b - start_id) + target_col] = M[input_row * dim + input_col];
}

__global__ void
forward_substitution_rectangular_a21(double *M, double *a11, double *a21_transpose, double *l21_transpose_from_fs, int dim, int b, int start_id)
{
	int i_index = (blockIdx.x * blockDim.x + threadIdx.x);
	int j_index = (blockIdx.y * blockDim.y + threadIdx.y);

	if (i_index >= (dim - b - start_id))
		return;

	/* for (int k = 0; k < dim2; k++) { // this is looping over columns of B matrix
		for (int i = 0; i < dim1; i++) {
			output[i][k] = B[i][k];
			for (int j = 0; j < i; j++) {
				output[i][k] = output[i][k] - A[i][j] * output[j][k];
			}
			output[i][k] = output[i][k] / A[i][i];
		}
	} */
	int k = i_index;
	// TODO experiment with #pragma unroll
	for (int i = 0; i < b; i++)
	{
		l21_transpose_from_fs[i * (dim - b - start_id) + k] = a21_transpose[i * (dim - b - start_id) + k];
		for (int j = 0; j < i; j++)
		{
			l21_transpose_from_fs[i * (dim - b - start_id) + k] -= a11[i * b + j] * l21_transpose_from_fs[j * (dim - b - start_id) + k];
		}
		l21_transpose_from_fs[i * (dim - b - start_id) + k] /= a11[i * b + i];
		
		//Updating M too!	
		M[ (start_id + b + k) * dim + start_id + i ] = l21_transpose_from_fs[i * (dim - b - start_id) + k];
	}
}

__global__ void
check_l21_kernel(double *M1, double *M2, double* targetoutput, int d1, int d2, int d3){
	
	double totaldiff = 0.0, diff = 0;
	for(int i = 0; i < d1; i++){
		for(int j = 0; j < d3 ;j++){ 
			double tempval = 0.0;
			for(int k = 0; k < d2; k++){
				//tempval += M1[i][k] + M2[k][j];
				tempval += M1[i*d2 + k] * M2[k * d3 + j];
			}
			//diff = tempval - targetoutput[i][j];
			diff = tempval - targetoutput[i * d3 + j];

			totaldiff += diff * diff;
			//printf("Diff = %lf\n", diff);
		}
	}
	printf("The error for l21_transpose_from_fs is %lf\n", totaldiff);
}
__global__ void
singlethread_temp_matmult_kernel(double *M1, double *M2, double* targetoutput, int d1, int d2, int d3){	
	for(int i = 0; i < d1; i++){
		for(int j = 0; j < d3 ;j++){ 
			double tempval = 0.0;
			for(int k = 0; k < d2; k++){
				tempval += M1[i*d2 + k] * M2[k * d3 + j];
			}
			targetoutput[i * d3 + j] = tempval;
		}
	}
}

__global__ void
generic_matrix_transpose(double *input, double *output, int d1, int d2){
	int i_index = (blockIdx.x * blockDim.x + threadIdx.x);
	int j_index = (blockIdx.y * blockDim.y + threadIdx.y);

	if (i_index >= d1 * d2)
		return;

	// DESIRED: output[j][i] = input[i][j];
	//	    output: d2 x d1
	// 	    input : d1 x d2
	
	// int input_row = index / d2;
	// int input_col = index % d2;
	output[ (i_index % d2) * d1 + (i_index / d2)]  = input[i_index];
}

__global__ void matrixmultiply_noshare(double *a, int rowsA, int colsA, double *b, int rowsB, int colsB, double *c)
{

	long long int col = blockIdx.y * blockDim.y + threadIdx.y;
	long long int row = blockIdx.x * blockDim.x + threadIdx.x;

	if (row >= rowsA || col >= colsB)
		return;

	double sum = 0.0;
	for (int i = 0; i < colsA; i++)
	{
		sum += a[row * colsA + i] * b[i * colsB + col]; 
	}

	c[row * colsB + col] = sum;
}

__global__ void offseted_elementwise_subtraction(double *input, int size, double *M, int dim, int b, int start_id){
	int i_index = (blockIdx.x * blockDim.x + threadIdx.x);
	int j_index = (blockIdx.y * blockDim.y + threadIdx.y);

	if (i_index >= size * size)
		return;

	// int input_row = i_index / size;
	// int input_col = i_index % size;
	// we want M[ input_row + start_id + b, input_col + start_id + b ] -= input[i_index];
	
	int input_row = i_index / size;
	int input_col = i_index % size;
	M[ (input_row + start_id + b) * dim + (input_col + start_id + b) ] -= input[i_index];
	
}

__global__ void
get_determinant_from_L(double *M, int dim, double *log_det)
{
	// single thread

	double ans = 0.0;
	for (int i = 0; i < dim; i++)
		ans += log(M[i * dim + i]);
	ans *= 2;
	*log_det = ans;
	printf("Determinant is %lf\n", ans);
}

__global__ void
elementwise_matrix_mult(double *mat1, double *mat2, double *mat3, int rows, int cols)
{
	int i_index = (blockIdx.x * blockDim.x + threadIdx.x);
	int j_index = (blockIdx.y * blockDim.y + threadIdx.y);

	int target_row, target_col;
	double dot_product = 0.0;

	target_row = i_index / cols;
	target_col = i_index % cols;

	if (target_row >= rows || target_col >= cols)
		return;

	mat3[target_row * cols + target_col] = mat1[target_row * cols + target_col] * mat2[target_row * cols + target_col];
}

__global__ void
compute_K_train(double *M, double *K_output, double *loghyper, int n, int dim) {
	int i_index = (blockIdx.x * blockDim.x + threadIdx.x);
	int j_index = (blockIdx.y * blockDim.y + threadIdx.y);

	if(i_index >= n * n) return;

	double ell_sq = exp(loghyper[0] * 2); //l^2 after coverting back from the log form
	double signal_var = exp(loghyper[1] * 2); // signal variance
	double noise_var = exp(loghyper[2] * 2); //noise variance

	int M_row, M_col;
	double dot_product = 0.0;

	M_row = i_index / n;
	M_col = i_index % n;

	if (M_row < M_col) // upper triangular bye bye
		return;

	if (M_row == M_col){
		K_output[M_row * n + M_col] = signal_var +  noise_var;
		return;
	}

	for (int i = 0; i < dim; i++)
		dot_product += (M[M_row * dim + i] - M[M_col * dim + i]) * (M[M_row * dim + i] - M[M_col * dim + i]);

	dot_product = signal_var * exp(-dot_product * 0.5 / ell_sq);

	K_output[M_row * n + M_col] = K_output[M_col * n + M_row] = dot_product;
}

__global__ void
compute_squared_distances(double *M, double *compute_squared_distances_matrix, double c, int n, int dim) {
	int i_index = (blockIdx.x * blockDim.x + threadIdx.x);
	int j_index = (blockIdx.y * blockDim.y + threadIdx.y);
	
	if(i_index >= n * n) return;

	int M_row, M_col;

	M_row = i_index / n;
	M_col = i_index % n;

	if (M_row < M_col) // upper triangular bye bye
		return;
	
	if (M_row == M_col)
	{
		compute_squared_distances_matrix[M_row * n + M_col] = 0.0;
		return;
	}
	double dot_product = 0.0;
	for (int i = 0; i < dim; i++)
		dot_product += (M[M_row * dim + i] - M[M_col * dim + i]) * (M[M_row * dim + i] - M[M_col * dim + i]);

	compute_squared_distances_matrix[M_row * n + M_col] = compute_squared_distances_matrix[M_col * n + M_row] = dot_product / c;
}

void get_symmetric_matrix_1d(double *M, double **matrix1, double **matrix2, int dim) {

	srand(time(NULL));
	int setter = 1;
	for (int i = 0; i < dim; i++)
	{
		for (int j = 0; j < dim; j++){
			matrix1[i][j] = rand() % 100 + 1;
			matrix2[j][i] = matrix1[i][j];
		}
	}
	for (int i = 0; i < dim; i++){
		for(int j = 0; j < dim; j++){
	M[i * dim + j ] = 0.0;
			for(int k = 0; k < dim; k++){
				M[i * dim + j] += matrix1[i][k]*matrix2[k][j];
			}
		}
	}
}

void init_and_print()
{
	int deviceCount = 0;
	bool isFastGPU = false;
	std::string name;
	hipError_t err = hipGetDeviceCount(&deviceCount);

	printf("---------------------------------------------------------\n");
	printf("Initializing CUDA for cuGP\n");
	printf("Found %d CUDA devices\n", deviceCount);

	for (int i=0; i<deviceCount; i++) {
		hipDeviceProp_t deviceProps;
		hipGetDeviceProperties(&deviceProps, i);
		name = deviceProps.name;
		if (name.compare("GeForce GTX 480") == 0
				|| name.compare("GeForce GTX 670") == 0
				|| name.compare("GeForce GTX 780") == 0)
		{
			isFastGPU = true;
		}

		printf("Device %d: %s\n", i, deviceProps.name);
		printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
		printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
		printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
	}
	printf("---------------------------------------------------------\n");
	if (!isFastGPU)
	{
		printf("WARNING: "
				"You're not running on a fast GPU, please consider using "
				"NVIDIA GTX 480, 670 or 780.\n");
		printf("---------------------------------------------------------\n");
	}
}

void setup_cholesky(int dim, int b)
{
	

	cudacall(hipMalloc(&M, sizeof(double) * dim * dim));
	cudacall(hipMalloc(&mt, sizeof(double)*dim * dim));
	cudacall(hipMalloc(&mt_transpose, sizeof(double)*dim * dim));
	cudacall(hipMemcpy(mt, temp_m, sizeof(double) * dim * dim, hipMemcpyHostToDevice));

	/*
	 * Now malloc the a11 matrix
	 */

	cudacall(hipMalloc(&a11, sizeof(double) * 4));

	/*
	 * Now malloc the a21_transpose matrix by overprovisioning. This can be of maximum size bx(dim-b). But, we allocate
	 * a bx(dim-b) vector even for the latter stages.
	 */

	 cudacall(hipMalloc(&a21_transpose, sizeof(double) * b * (dim - b)));
	 cudacall(hipMemset((void *)a21_transpose, 0, sizeof(double) * b * (dim - b)));

	/*
	 * Now malloc the l21_transpose_from_fs matrix to insert the output of forward substitution. Is retained here for generating a22.
	 */

	 cudacall(hipMalloc(&l21_transpose_from_fs, sizeof(double) * b * (dim - b)));
	 cudacall(hipMemset((void *)l21_transpose_from_fs, 0, sizeof(double) * b * (dim - b)));
	
	/*
	 * Now malloc the l21 matrix, which will be useful for populating a22 (via matrix mult).
	 */

	 cudacall(hipMalloc(&l21, sizeof(double) * b * (dim - b)));

	/*
	 * Now malloc the l22_temp matrix, which will be useful for elementwise subtraction for a22 (after matrix mult).
	 */

	 cudacall(hipMalloc(&l22_temp, sizeof(double) * (dim - b) * (dim - b)));
		
	
	 
	/*GlobalConstants params;
	  params.sceneName = sceneName;
	  params.numCircles = numCircles;
	  params.imageWidth = image->width;
	  params.imageHeight = image->height;
	  params.position = cudaDevicePosition;
	  params.velocity = cudaDeviceVelocity;
	  params.color = cudaDeviceColor;
	  params.radius = cudaDeviceRadius;
	  params.imageData = cudaDeviceImageData;

	  hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));

	// also need to copy over the noise lookup tables, so we can
	// implement noise on the GPU
	int* permX;
	int* permY;
	float* value1D;
	getNoiseTables(&permX, &permY, &value1D);
	hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseXPermutationTable), permX, sizeof(int) * 256);
	hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseYPermutationTable), permY, sizeof(int) * 256);
	hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoise1DValueTable), value1D, sizeof(float) * 256);

	// last, copy over the color table that's used by the shading // function for circles in the snowflake demo

	float lookupTable[COLOR_MAP_SIZE][3] = {
	{1.f, 1.f, 1.f},
	{1.f, 1.f, 1.f},
	{.8f, .9f, 1.f},
	{.8f, .9f, 1.f},
	{.8f, 0.8f, 1.f},
	};

	hipMemcpyToSymbol(HIP_SYMBOL(cuConstColorRamp), lookupTable, sizeof(float) * 3 * COLOR_MAP_SIZE);

	if(hipMalloc(&table, sizeof(int) * numCircles * image->width * image->height / 16 / 16) != hipSuccess )
	printf("The giant malloc failed.\n");

	if (hipMalloc(&counters, sizeof(int) * image->height * image->width / 16 / 16) != hipSuccess)
	printf("The counter malloc fialed.\n");

	if (hipMalloc(&flagarray, sizeof(int) * numCircles) != hipSuccess)
	printf("The flag array malloc failed.\n");

	hipMemset((void *)flagarray, 0, sizeof(int) * numCircles); */
}

__inline__ int upit(int x, int y) {
	return (x + y - 1) / y;
}


void check_cholesky(double *M1, double* targetoutput, int d){
	double diff = 0.0, totaldiff = 0.0;	
	for(int i = 0; i < d; i++){
		for(int j = 0; j < d ;j++){ 
			double tempval = 0.0;
			for(int k = 0; k < d; k++){
				//tempval += M1[i*d + k] * M2[k * d + j];
				tempval += M1[i*d + k] * M1[j * d + k];
			}
			diff = tempval - targetoutput[i * d + j];
			totaldiff += abs(diff);
		}
	}
	printf("FINAL ERROR = %lf\n", totaldiff);
}

void get_input(int dim){

	double **m1, **m2;

	temp_m = new double[dim * dim];

	m1 = new double *[dim];
	m2 = new double *[dim];

	for (int i = 0; i < dim; i++)
	{
		m1[i] = new double[dim];
		m2[i] = new double[dim];
	}

	get_symmetric_matrix_1d(temp_m, m1, m2, dim);
	
	printf("Abhi input hua\n");
	/*
	printf("Generated matrix in host is \n");
	for (int i = 0; i < dim; i++)
	{
		for (int j = 0; j < dim; j++)
		{
			printf("%lf ", temp_m[i * dim + j]);
		}
		printf("\n");
	}
	*/
	
	for(int i = 0 ; i < dim ; i++){
		delete m1[i];
		delete m2[i];
	}
	delete m1;
	delete m2;

}
void initialize_random(int dim){
	temp_m = new double[dim * dim];
	srand(time(NULL));
        for (int i = 0; i < dim; i++)
        {
                for (int j = 0; j < dim; j++){
                        temp_m[i*dim + j] = ((double) rand() / (RAND_MAX));
                        //temp_m[i*dim + j] = rand() % 10;
		}
	}
	
}

void run_kernel_cholesky(int dim)
{
	int start_id, b;
	int threads_per_block;
	int number_of_blocks;
	int num_iters;

	double startime, endtime ;
	start_id = 0;
//	dim = 5000;
	b = 2;
	start_id = 0;

	init_and_print();
//	get_input(dim);
	initialize_random(dim);
	printf("okay random bhar gaya\n");
	setup_cholesky(dim, b);
	
	// Input generation
	//	1. taking transpose of mt in mt_transpose
	threads_per_block = 1024;
	number_of_blocks = upit(dim * dim, threads_per_block);
	generic_matrix_transpose<<<number_of_blocks, threads_per_block>>>(mt, mt_transpose, dim, dim);
	hipDeviceSynchronize();
	printf("ab jakar transpose hua\n");
	/*
	print_matrix_kernel<<<1, 1>>>(mt, dim, dim);
	hipDeviceSynchronize();
	print_matrix_kernel<<<1, 1>>>(mt_transpose, dim, dim);
	hipDeviceSynchronize();
	*/
	
	startime = CycleTimer::currentSeconds();	
	dim3 blockDimTemp(32,32);
	dim3 gridDimTemp( upit(dim, blockDimTemp.x), upit(dim, blockDimTemp.y));
	//matrixmultiply_noshare(double *a, int rowsA, int colsA, double *b, int rowsB, int colsB, double *c)
	matrixmultiply_noshare<<<gridDimTemp, blockDimTemp >>>(mt, dim, dim,  mt_transpose, dim, dim, M);
	hipDeviceSynchronize();
	endtime = CycleTimer::currentSeconds();	
	printf("Now multiplication got over, total time taken for dim = %d, is %lf\n", dim, endtime - startime);

	// Now copying the symmetric matrix from CUDA to host
	orig_sym = new double[dim * dim];
	cudacall(hipMemcpy(orig_sym, M,  sizeof(double) * dim * dim, hipMemcpyDeviceToHost));
	
	printf("Host me aya kyaa??\n");
	
	// WRITING TO FILE
	/*
	std::ofstream out(filename);
	for(int i = 0; i < dim ; i++){
		for(int j = 0; j < dim ; j++){
			out << orig_sym[i*dim + j] << " ";
	//		printf("%lf ", orig_sym[i*dim + j]);
		}
		out << "\n";
	//	printf("\n");
	}
	out.close();
	*/	

	startime = CycleTimer::currentSeconds();
	num_iters = dim / b;
	for (int i = 0; i < num_iters; i++)
	{
		hardcoded_cholesky_2x2<<<1, 1>>>(M, a11, dim, b, start_id);
		hipDeviceSynchronize();

		if (i == num_iters - 1)
			break;

		// TODO optimize a21_transpose, by bypassing it perhaps? Can avoid transpose and manipulate indices inside next kernel
		threads_per_block = 512;
		number_of_blocks = upit((dim - b - start_id) * b, threads_per_block);
		take_a21_transpose<<<number_of_blocks, threads_per_block>>>(M, a21_transpose, dim, b, start_id);
		hipDeviceSynchronize();

		threads_per_block = 512;
		number_of_blocks = upit((dim - b - start_id), threads_per_block);
		forward_substitution_rectangular_a21<<<number_of_blocks, threads_per_block>>>(M, a11, a21_transpose, l21_transpose_from_fs, dim, b, start_id);
		hipDeviceSynchronize();

	//	printf("Printing l21_transpose_from_fs\n");
	//	print_matrix_kernel<<<1, 1>>>(l21_transpose_from_fs, b, dim - b - start_id);
	//	hipDeviceSynchronize();

		/*		
		printf("\n\n");
		printf(" ---------------------------------------- \n");	
		print_matrix_kernel<<<1, 1>>>(a11, b, b);
		hipDeviceSynchronize();
		printf(" ---------------------------------------- \n");
		print_matrix_kernel<<<1,1>>>(a21_transpose, b, dim - b - start_id);
		hipDeviceSynchronize();
		printf(" ---------------------------------------- \n");
		singlethread_temp_matmult_kernel<<<1, 1>>>(a11, a21_transpose, l21_transpose_from_fs, b, b, dim - b - start_id);
		hipDeviceSynchronize();
		print_matrix_kernel<<<1,1>>>(l21_transpose_from_fs, b, dim - b - start_id);
		hipDeviceSynchronize();
		printf("\n\n");
		*/
			
		//printf("\nNow printing entire M matrix\n");
		//print_matrix_kernel<<<1, 1>>>(M, dim, dim);
		//hipDeviceSynchronize();
		
		// TODO: Can include this tranpose in the forward_substitution_rectangular_a22 call!!!!
		// Now taking transpose of l21_transpose_from_fs
		 
		threads_per_block = 512;
		number_of_blocks = upit((dim - b - start_id) * b, threads_per_block);
		generic_matrix_transpose<<<number_of_blocks, threads_per_block>>>(l21_transpose_from_fs, l21, b, dim - b - start_id);
		hipDeviceSynchronize();
		
//		printf("\nNow checking the transpose => \n");	
//		print_matrix_kernel<<<1,1>>>(l21, dim - b - start_id, b);
//		hipDeviceSynchronize();
//		printf("Checking the l21_transpose_from_fs matrix\n");
//		check_l21_kernel<<<1, 1>>>(a11, l21_transpose_from_fs, a21_transpose, b, b, dim - b - start_id);
//		hipDeviceSynchronize();

		//matrixmultiply_noshare<<<(double *a, int rowsA, int colsA, double *b, int rowsB, int colsB, double *c)
		int rowA = (dim - b - start_id) , colA = b, rowB = b , colB = (dim - b - start_id) ;
		dim3 blockDim(32,32);
		dim3 gridDim( upit(colB, blockDim.x), upit(rowA, blockDim.y));
		matrixmultiply_noshare<<<gridDim, blockDim >>>(l21, (dim - b - start_id), b,  l21_transpose_from_fs, b, dim - b - start_id, l22_temp);
		hipDeviceSynchronize();

		threads_per_block = 512;
		number_of_blocks = upit((dim - b - start_id) * (dim - b - start_id), threads_per_block);
		offseted_elementwise_subtraction<<<number_of_blocks, threads_per_block >>>(l22_temp, dim - b - start_id, M, dim, b, start_id);
		hipDeviceSynchronize();

		start_id += b;
	}
	// Fire a kernel for making upper-triangular as 0.0
	threads_per_block = 512;
	number_of_blocks = upit( (dim * dim), threads_per_block);
	set_upper_zero<<<number_of_blocks, threads_per_block>>>(M, dim);
	hipDeviceSynchronize();
	endtime = CycleTimer::currentSeconds();	
	printf("Totat time taken = %lf s\n", endtime - startime);	
	// Now checking!
	
	double *finalans = new double[dim * dim];
	cudacall(hipMemcpy(finalans, M,  sizeof(double) * dim * dim, hipMemcpyDeviceToHost));
	check_cholesky(finalans, orig_sym, dim);	

	/*for(int i = 0; i < dim ; i++){
		for(int j = 0; j < dim ; j++){
			printf("%lf ", finalans[i*dim + j]);
		}
		printf("\n");
	}*/
	
}

void generate_random_vector(double *b, int dim){
	for(int i = 0 ; i < dim ; i++){
		b[i] = rand() % 10;	
	}
}

void get_cholesky(int n)
{
	int start_id, b;
	int threads_per_block;
	int number_of_blocks;
	int num_iters;
	double startime, endtime;
	int dim = n;

	start_id = 0;
	b = 2;

	startime = CycleTimer::currentSeconds();	

	num_iters = n / b;
	for (int i = 0; i < num_iters; i++)
	{
		hardcoded_cholesky_2x2<<<1, 1>>>(M, a11, dim, b, start_id);
		hipDeviceSynchronize();

		if (i == num_iters - 1)
			break;

		// TODO optimize a21_transpose, by bypassing it perhaps? Can avoid transpose and manipulate indices inside next kernel
		threads_per_block = 512;
		number_of_blocks = upit((dim - b - start_id) * b, threads_per_block);
		take_a21_transpose<<<number_of_blocks, threads_per_block>>>(M, a21_transpose, dim, b, start_id);
		hipDeviceSynchronize();

		threads_per_block = 512;
		number_of_blocks = upit((dim - b - start_id), threads_per_block);
		forward_substitution_rectangular_a21<<<number_of_blocks, threads_per_block>>>(M, a11, a21_transpose, l21_transpose_from_fs, dim, b, start_id);
		hipDeviceSynchronize();

		threads_per_block = 512;
		number_of_blocks = upit((dim - b - start_id) * b, threads_per_block);
		generic_matrix_transpose<<<number_of_blocks, threads_per_block>>>(l21_transpose_from_fs, l21, b, dim - b - start_id);
		hipDeviceSynchronize();
		
		//matrixmultiply_noshare<<<(double *a, int rowsA, int colsA, double *b, int rowsB, int colsB, double *c)
		int rowA = (dim - b - start_id) , colA = b, rowB = b , colB = (dim - b - start_id) ;
		dim3 blockDim(32,32);
		dim3 gridDim( upit(colB, blockDim.x), upit(rowA, blockDim.y));
		matrixmultiply_noshare<<<gridDim, blockDim >>>(l21, (dim - b - start_id), b,  l21_transpose_from_fs, b, dim - b - start_id, l22_temp);
		hipDeviceSynchronize();

		threads_per_block = 512;
		number_of_blocks = upit((dim - b - start_id) * (dim - b - start_id), threads_per_block);
		offseted_elementwise_subtraction<<<number_of_blocks, threads_per_block >>>(l22_temp, dim - b - start_id, M, dim, b, start_id);
		hipDeviceSynchronize();

		start_id += b;
	}
	// Fire a kernel for making upper-triangular as 0.0
	threads_per_block = 512;
	number_of_blocks = upit( (dim * dim), threads_per_block);
	set_upper_zero<<<number_of_blocks, threads_per_block>>>(M, dim);
	hipDeviceSynchronize();
	endtime = CycleTimer::currentSeconds();	
	printf("Totat time taken = %lf s\n", endtime - startime);	
	// Now checking!
	
	double *finalans = new double[dim * dim];
	cudacall(hipMemcpy(finalans, M,  sizeof(double) * dim * dim, hipMemcpyDeviceToHost));
	check_cholesky(finalans, orig_sym, dim);	
}

void compute_chol_get_mul_and_det()
{
	// get_cholesky(); // set of kernels

	// compute_determinant(); // kernel

	// matrix_transpose(); // kernel

	// forward_solve_vector(); // kernel Ly=b

	// backward_solve_vector(); // kernel Ux=y

	// compute_product(); // kernel
}

__global__ void compute_log_likelihood()
{
	// compute_K_train(); // kernel

	// compute_chol_get_mul_and_det(); // set of kernels
	
	// evaluate_and_store_log_likelihood(); // kernel, or can be clubbed somewhere
}

void compute_K_inverse()
{
	// make_identity(); // kernel, or do once, and store

	// get_cholesky(); // set of kernels

	// matrix_forward_substitution(); // kernel

	// matrix_transpose(); // kernel

	// matrix_backward_substitution(); // kernel
}

void vector_Kinvy_using_cholesky()
{
	// get_cholesky(); // set of kernels

	// matrix_transpose();

	// forward_solve_vector();

	// backward_solve_vector();
}

void compute_gradient_log_hyperparams()
{
	// compute_K_train(); // kernel - can reuse earlier matrix?

	// compute_squared_distance(); // kernel

	// elementwise_matrix_mult(); // kernel

	// compute_K_inverse(); // set of kernels

	// vector_Kinvy_using_cholesky(); // set of kernels

	// get_outer_product(); // kernel

	// subtract_matrices(); // kernel

	// update_log_hyperparams(); // kernel
}

void run_kernel(){
	//Now checking matrix 

	printf("Okay called at least\n");
	int threads_per_block;
	int number_of_blocks;
	
	FILE *input_file, *label_file;
	input_file = fopen(INPUT_FILE, "r");
	int n, dim;

	fscanf(input_file, "%d%d", &n, &dim);

	double *X; //input dataset in host!
	double *lh_host = new double[3];
	for(int i = 0 ; i < 3 ; i++){
		lh_host[i] = 0.5;	
	}
        X = new double[n*dim];

        for (int i = 0; i < n; i++)
                for (int j = 0; j < dim; j++)
                        fscanf(input_file, "%lf", &X[i*dim + j]);
	
	double *inputdata;
	double *loghyper;
	double *K_output; //for storing the n x n matrix

	cudacall(hipMalloc(&inputdata, sizeof(double) * dim * n));
	cudacall(hipMemcpy(inputdata, X, sizeof(double) * dim * n , hipMemcpyHostToDevice));	
	
	cudacall(hipMalloc(&loghyper, sizeof(double) * 3));
	cudacall(hipMemcpy(loghyper, lh_host, sizeof(double) * 3 , hipMemcpyHostToDevice));	
	
	cudacall(hipMalloc(&K_output, sizeof(double) * n * n));	
	printf("n = %d, dim = %d\n", n, dim);	
	threads_per_block = 512;
	number_of_blocks = upit( (n * n), threads_per_block);

	compute_K_train<<<number_of_blocks, threads_per_block >>>(inputdata, K_output, loghyper, n,  dim);	
	hipDeviceSynchronize();
	print_matrix_kernel<<<1,1>>>(K_output, n, n);
	hipDeviceSynchronize();

	printf("\nNow printing the squared distance matrix\n");	
			
	double c = exp(lh_host[0] * 2); 
	threads_per_block = 512;
	number_of_blocks = upit( (n * n), threads_per_block);
	compute_squared_distances<<<number_of_blocks, threads_per_block>>>(inputdata,  K_output,  c,  n, dim);
	hipDeviceSynchronize();
	print_matrix_kernel<<<1,1>>>(K_output, n, n);
	hipDeviceSynchronize();
	return ;
	printf("Abey yahan toh aya\n");	
	int N = 10;	 //Total number of training samples
	run_kernel_cholesky(N);
	printf("Call to cholesky khatam hua\n");	


	//NOTE: M will now have a lower triangular matrix
	print_matrix_kernel<<<1,1>>>(M, N, N);
	hipDeviceSynchronize();

	double *mat1, *mat2, *mat3;
	double mat1_host[16], mat2_host[16], mat3_host[16];

	N = 16; // 4x4 matrices
	cudacall(hipMalloc(&mat1, sizeof(double) * N));
	cudacall(hipMalloc(&mat2, sizeof(double) * N));
	cudacall(hipMalloc(&mat3, sizeof(double) * N));

	for (int i = 0; i < 16; i++)
		mat1_host[i] = i;

	for (int j = 16; j < 32; j++)
		mat2_host[j - 16] = j;

	cudacall(hipMemcpy(mat1, mat1_host, sizeof(double) * N , hipMemcpyHostToDevice));	
	cudacall(hipMemcpy(mat2, mat2_host, sizeof(double) * N , hipMemcpyHostToDevice));	

	printf("mat1:\n");
	print_matrix_kernel<<<1, 1>>>(mat1, 4, 4);
	hipDeviceSynchronize();

	printf("mat2:\n");
	print_matrix_kernel<<<1, 1>>>(mat2, 4, 4);
	hipDeviceSynchronize();

	elementwise_matrix_mult<<<1, 256>>>(mat1, mat2, mat3, 4, 4);
	hipDeviceSynchronize();

	printf("mat3:\n");
	print_matrix_kernel<<<1, 1>>>(mat3, 4, 4);
	hipDeviceSynchronize();

	return ;

	double *log_det;
	cudacall(hipMalloc(&log_det, sizeof(double) * 1));
	get_determinant_from_L<<<1, 1>>>(M, N, log_det);
	hipDeviceSynchronize();
	return;	

	// FORWARD SUBSTITUTION 
	//	
	//	generating random targets!!	
	double *b = new double[N];

	double *labels_vec; //This is the array with the target values in the dataset, it is vector for CUDA
			    // Next, we will have to load the values from the file instead of copying from b	

	double *fsvec; // This is the array that will contain the result of ForwardSubstitutionVector call
	
	generate_random_vector(b, N);	
	//	Allocating appropriate memory chunks.
	cudacall(hipMalloc(&labels_vec, sizeof(double) * N));
	cudacall(hipMemcpy(labels_vec, b, sizeof(double) * N , hipMemcpyHostToDevice));	
	cudacall(hipMalloc(&fsvec, sizeof(double) * N));
	
	threads_per_block = 512;
	number_of_blocks = upit(N, threads_per_block);
	forward_substitution_vector<<<1, 1>>>(M, labels_vec, fsvec, N);
	hipDeviceSynchronize();

	// Checking for forwardSubstitutionVector
	check_forward_sub_vector<<<1, 1>>>(M, fsvec, labels_vec, N);
	hipDeviceSynchronize();

	
	// BACKWARD SUBSTITUTION
	
	double *bsvec;
	//	Allocating appropriate memory chunks
	cudacall(hipMalloc(&bsvec, sizeof(double) * N));
	backward_substitution_vector<<<1, 1>>>(M, fsvec, bsvec, N); // Will use M.transpose() inside!!
	hipDeviceSynchronize();
	
	// Checking for backwardSubstitutionVector
	check_backward_sub_vector<<<1, 1>>>(M, bsvec, fsvec, N);
	hipDeviceSynchronize();
			
}
