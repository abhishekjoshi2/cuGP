#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

double *M;
double *a11;
double *a21_transpose;
double *l21_from_fs;

#define cudacall(call) \
{ \
	hipError_t err = (call);                                                                                               \
	if(hipSuccess != err)                                                                                                  \
	{                                                                                                                       \
		fprintf(stderr,"CUDA Error:\nFile = %s\nLine = %d\nReason = %s\n", __FILE__, __LINE__, hipGetErrorString(err));    \
		hipDeviceReset();                                                                                                  \
		exit(EXIT_FAILURE);                                                                                                 \
	}                                                                                                                       \
} \


	__global__ void
hardcoded_cholesky_1x1(double *M, double *a11, int dim, int b, int start_id)
{
	// TODO
	/* M[idx][idx] = sqrt(M[idx][idx]);
	a11[0][0] = M[idx][idx]; */
}

	__global__ void
hardcoded_cholesky_2x2(double *M, double *a11, int dim, int b, int start_id)
{
	int i_index = (blockIdx.x * blockDim.x + threadIdx.x);
	int j_index = (blockIdx.y * blockDim.y + threadIdx.y);

	printf("In kernel\n");
	printf("dim is %d, i_index is %d, j_index is %d, b is %d, start_id is %d\n", dim, i_index, j_index, b, start_id);

	printf("Now the 2x2 matrix is:\n");
	for (int i = 0; i < b; i++)
	{
		for (int j = 0; j < b; j++)
		{
			printf("%lf ", M[(i + start_id) * dim + j + start_id]);
		}
		printf("\n");
	}

	a11[0] = M[start_id * dim + start_id] = sqrt(M[start_id * dim + start_id]);
	a11[1] = M[start_id * dim + start_id + 1] = 0.0;
	a11[2] = M[(start_id + 1) * dim + start_id] = M[(start_id + 1) * dim + start_id] / M[start_id * dim + start_id];
	a11[3] = M[(start_id + 1) * dim + start_id + 1] = sqrt(M[(start_id + 1) * dim + start_id + 1] - M[(start_id + 1) * dim + start_id] * M[(start_id + 1) * dim + start_id]);

	printf("printing a11 matrix\n");
	printf("%lf %lf %lf %lf\n", a11[0], a11[1], a11[2], a11[3]);
}

__global__ void
print_matrix_kernel(double *arr, int dim1, int dim2)
{
	printf("Printing matrix:\n");
	for (int i = 0; i < dim1; i++)
	{
		for (int j = 0; j < dim2; j++)
		{
			printf("%lf ", arr[i * dim2 + j]);
		}
		printf("\n");
	}
}

__global__ void
take_a21_transpose(double *M, double *a21_transpose, int dim, int b, int start_id) {
	int i_index = (blockIdx.x * blockDim.x + threadIdx.x);
	int j_index = (blockIdx.y * blockDim.y + threadIdx.y);

	if (i_index >= (dim - b - start_id) * b)
		return;

	printf("In a21_transpose, i_index is %d, j_index is %d\n", i_index, j_index);

	int input_row, input_col, target_row, target_col, row_offset_by_thread, col_offset_by_thread;

	// TODO replace i_index by a generic construct because it may involve blocks and grids
	row_offset_by_thread = i_index / b;
	input_row = start_id + b + row_offset_by_thread;

	col_offset_by_thread = i_index % b;
	input_col = start_id + col_offset_by_thread;

	target_row = i_index % b;
	target_col = i_index / b;

	a21_transpose[target_row * (dim - b - start_id) + target_col] = M[input_row * dim + input_col];
}

__global__ void
forward_substitution_rectangular_a21(double *a11, double *a21_transpose, double *l21_from_fs, int dim, int b, int start_id)
{
	int i_index = (blockIdx.x * blockDim.x + threadIdx.x);
	int j_index = (blockIdx.y * blockDim.y + threadIdx.y);

	if (i_index >= (dim - b - start_id))
		return;

	/* for (int k = 0; k < dim2; k++) { // this is looping over columns of B matrix
		for (int i = 0; i < dim1; i++) {
			output[i][k] = B[i][k];
			for (int j = 0; j < i; j++) {
				output[i][k] = output[i][k] - A[i][j] * output[j][k];
			}
			output[i][k] = output[i][k] / A[i][i];
		}
	} */
	int k = i_index;
	// TODO experiment with #pragma unroll
	for (int i = 0; i < b; i++)
	{
		l21_from_fs[i * (dim - b - start_id) + k] = a21_transpose[i * (dim - b - start_id) + k];
		for (int j = 0; j < i; j++)
		{
			l21_from_fs[i * (dim - b - start_id) + k] -= a11[i * b + j] * l21_from_fs[j * (dim - b - start_id) + k];
		}
		l21_from_fs[i * (dim - b - start_id) + k] /= a11[i * b + i];
	}
}

__global__ void
check_l21_kernel(double *M1, double *M2, double* targetoutput, int d1, int d2, int d3){
	
	double totaldiff = 0.0, diff = 0;
	for(int i = 0; i < d1; i++){
		for(int j = 0; j < d3 ;j++){ 
			double tempval = 0.0;
			for(int k = 0; k < d2; k++){
				//tempval += M1[i][k] + M2[k][j];
				tempval += M1[i*d2 + k] * M2[k * d3 + j];
			}
			//diff = tempval - targetoutput[i][j];
			diff = tempval - targetoutput[i * d3 + j];

			totaldiff += diff * diff;
			//printf("Diff = %lf\n", diff);
		}
	}
	printf("The error for l21_from_fs is %lf\n", totaldiff);
}
__global__ void
matrixmultiply_kernel(double *M1, double *M2, double* targetoutput, int d1, int d2, int d3){
	
	for(int i = 0; i < d1; i++){
		for(int j = 0; j < d3 ;j++){ 
			double tempval = 0.0;
			for(int k = 0; k < d2; k++){
				tempval += M1[i*d2 + k] * M2[k * d3 + j];
			}
			targetoutput[i * d3 + j] = tempval;

		}
	}
}


__global__ void kernelMatMult(double *a, int rowsA, int colsA,
		double *b, int rowsB, int colsB, double *c)
{

	int col = blockIdx.y * blockDim.y + threadIdx.y;
	int row = blockIdx.x * blockDim.x + threadIdx.x;

	if (row >= rowsA || col >= colsB)
		return;

	float sum = 0.0f;
	for (int i = 0; i < colsA; i++)
	{
		sum += a[row * colsA + i] * b[i * colsB + col]; 
	}

	c[row * colsB + col] = sum;
}



void get_symmetric_matrix_1d(double *M, double **matrix1, double **matrix2, int dim) {

	srand(time(NULL));
	for (int i = 0; i < dim; i++)
	{
		for (int j = 0; j < dim; j++){
			matrix1[i][j] = rand() % 100 + 1;
			matrix2[j][i] = matrix1[i][j];
		}
	}

	for (int i = 0; i < dim; i++){
		for(int j = 0; j < dim; j++){
			for(int k = 0; k < dim; k++){
				M[i * dim + j] += matrix1[i][k]*matrix2[k][j];
			}
		}
	}
}

void init_and_print()
{
	int deviceCount = 0;
	bool isFastGPU = false;
	std::string name;
	hipError_t err = hipGetDeviceCount(&deviceCount);

	printf("---------------------------------------------------------\n");
	printf("Initializing CUDA for cuGP\n");
	printf("Found %d CUDA devices\n", deviceCount);

	for (int i=0; i<deviceCount; i++) {
		hipDeviceProp_t deviceProps;
		hipGetDeviceProperties(&deviceProps, i);
		name = deviceProps.name;
		if (name.compare("GeForce GTX 480") == 0
				|| name.compare("GeForce GTX 670") == 0
				|| name.compare("GeForce GTX 780") == 0)
		{
			isFastGPU = true;
		}

		printf("Device %d: %s\n", i, deviceProps.name);
		printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
		printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
		printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
	}
	printf("---------------------------------------------------------\n");
	if (!isFastGPU)
	{
		printf("WARNING: "
				"You're not running on a fast GPU, please consider using "
				"NVIDIA GTX 480, 670 or 780.\n");
		printf("---------------------------------------------------------\n");
	}
}

void setup(int dim, int b)
{
	double *temp_m, **m1, **m2;

	/*
	 * First generate the M matrix
	 */
	temp_m = new double[dim * dim];

	m1 = new double *[dim];
	m2 = new double *[dim];

	for (int i = 0; i < dim; i++)
	{
		m1[i] = new double[dim];
		m2[i] = new double[dim];
	}

	get_symmetric_matrix_1d(temp_m, m1, m2, dim);

	printf("Generated matrix in host is \n");
	for (int i = 0; i < dim; i++)
	{
		for (int j = 0; j < dim; j++)
		{
			printf("%lf ", temp_m[i * dim + j]);
		}
		printf("\n");
	}

	cudacall(hipMalloc(&M, sizeof(double) * dim * dim));
	cudacall(hipMemcpy(M, temp_m, sizeof(double) * dim * dim, hipMemcpyHostToDevice));

	/*
	 * Now malloc the a11 matrix
	 */

	cudacall(hipMalloc(&a11, sizeof(double) * 4));

	/*
	 * Now malloc the a21_transpose matrix by overprovisioning. This can be of maximum size bx(dim-b). But, we allocate
	 * a bx(dim-b) vector even for the latter stages.
	 */

	 cudacall(hipMalloc(&a21_transpose, sizeof(double) * b * (dim - b)));
	 cudacall(hipMemset((void *)a21_transpose, 0, sizeof(double) * b * (dim - b)));

	/*
	 * Now malloc the l21_from_fs matrix to insert the output of forward substitution. Is retained here for generating a22.
	 */

	 cudacall(hipMalloc(&l21_from_fs, sizeof(double) * b * (dim - b)));
	 cudacall(hipMemset((void *)l21_from_fs, 0, sizeof(double) * b * (dim - b)));

	/*GlobalConstants params;
	  params.sceneName = sceneName;
	  params.numCircles = numCircles;
	  params.imageWidth = image->width;
	  params.imageHeight = image->height;
	  params.position = cudaDevicePosition;
	  params.velocity = cudaDeviceVelocity;
	  params.color = cudaDeviceColor;
	  params.radius = cudaDeviceRadius;
	  params.imageData = cudaDeviceImageData;

	  hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));

	// also need to copy over the noise lookup tables, so we can
	// implement noise on the GPU
	int* permX;
	int* permY;
	float* value1D;
	getNoiseTables(&permX, &permY, &value1D);
	hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseXPermutationTable), permX, sizeof(int) * 256);
	hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseYPermutationTable), permY, sizeof(int) * 256);
	hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoise1DValueTable), value1D, sizeof(float) * 256);

	// last, copy over the color table that's used by the shading // function for circles in the snowflake demo

	float lookupTable[COLOR_MAP_SIZE][3] = {
	{1.f, 1.f, 1.f},
	{1.f, 1.f, 1.f},
	{.8f, .9f, 1.f},
	{.8f, .9f, 1.f},
	{.8f, 0.8f, 1.f},
	};

	hipMemcpyToSymbol(HIP_SYMBOL(cuConstColorRamp), lookupTable, sizeof(float) * 3 * COLOR_MAP_SIZE);

	if(hipMalloc(&table, sizeof(int) * numCircles * image->width * image->height / 16 / 16) != hipSuccess )
	printf("The giant malloc failed.\n");

	if (hipMalloc(&counters, sizeof(int) * image->height * image->width / 16 / 16) != hipSuccess)
	printf("The counter malloc fialed.\n");

	if (hipMalloc(&flagarray, sizeof(int) * numCircles) != hipSuccess)
	printf("The flag array malloc failed.\n");

	hipMemset((void *)flagarray, 0, sizeof(int) * numCircles); */
}

__inline__ int upit(int x, int y) {
	return (x + y - 1) / y;
}

void run_kernel()
{
	int dim, start_id, b;
	int threads_per_block;
	int number_of_blocks;
	int num_iters;

	start_id = 0;
	dim = 8;
	b = 2;
	start_id = 0;

	init_and_print();
	setup(dim, b);

	num_iters = dim / b;
	for (int i = 0; i < num_iters; i++)
	{
		printf("\n\n");
		printf("Iteration number is %d\n", i + 1);
		hardcoded_cholesky_2x2<<<1, 1>>>(M, a11, dim, b, start_id);
		hipDeviceSynchronize();

		if (i == num_iters - 1)
			break;

		// TODO optimize a21_transpose, by bypassing it perhaps? Can avoid transpose and manipulate indices inside next kernel
		threads_per_block = 256;
		number_of_blocks = upit((dim - b - start_id) * b, threads_per_block);
		printf("number_of_blocks is %d, threads_per_block is %d\n", number_of_blocks, threads_per_block);
		take_a21_transpose<<<number_of_blocks, threads_per_block>>>(M, a21_transpose, dim, b, start_id);
		hipDeviceSynchronize();

		threads_per_block = 256;
		number_of_blocks = upit((dim - b - start_id), threads_per_block);
		forward_substitution_rectangular_a21<<<number_of_blocks, threads_per_block>>>(a11, a21_transpose, l21_from_fs, dim, b, start_id);
		hipDeviceSynchronize();

		printf("Printing l21_from_fs\n");
		print_matrix_kernel<<<1, 1>>>(l21_from_fs, b, dim - b - start_id);
		hipDeviceSynchronize();

		/*		
		printf("\n\n");
		printf(" ---------------------------------------- \n");	
		print_matrix_kernel<<<1, 1>>>(a11, b, b);
		hipDeviceSynchronize();
		printf(" --------------------------------------- \n");
		print_matrix_kernel<<<1,1>>>(a21_transpose, b, dim - b - start_id);
		hipDeviceSynchronize();
		printf(" --------------------------------------- \n");
		matrixmultiply_kernel<<<1, 1>>>(a11, a21_transpose, l21_from_fs, b, b, dim - b - start_id);
		hipDeviceSynchronize();
		print_matrix_kernel<<<1,1>>>(l21_from_fs, b, dim - b - start_id);
		hipDeviceSynchronize();
		printf("\n\n");
		*/

		check_l21_kernel<<<1, 1>>>(a11, l21_from_fs, a21_transpose, b, b, dim - b - start_id);
		hipDeviceSynchronize();

		start_id += b;
	}
	printf("Kernel call done\n");
}
