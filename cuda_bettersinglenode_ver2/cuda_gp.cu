#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <driver_functions.h>
#include <cstdlib>
#include "../common/cycleTimer.h"
#include <fstream>
#include <thrust/inner_product.h>
#include <thrust/device_ptr.h>
#include "../cuda_src/Eigen/Dense"
#include "../cuda_src/Eigen/src/Core/util/DisableStupidWarnings.h"
#include <cmath>
#include<string>

#include <hipsolver.h>
#include <hipblas.h>
#include <hip/hip_runtime_api.h>

#include "Utilities.cuh"

#define BLOCK_SIZE 32

//For cublas
int *devInfo;
hipsolverHandle_t solver_handle;
hipblasHandle_t blas_handle;
hipblasStatus_t stat;

double *lowermat_inv_store;

bool correct = true;
double *temp_m; 

double *mt;
double *mt_transpose;

double *orig_sym;

// HOST gloval variables
double *lh_host; //Remember to free during cleanup


//For gradient
double *Ksqdist; // for squarred distances
double *matforell; //for gradient of ell
double *tempfsforkinv; //matrix for storing forward substitution result
double *Kinv; //inverse of K matrix
double *Kintact; //want to keep it
double *temp1dvec; //for storing Kinv*y
double *tempWmatrix; // (hard to explain, basically a intermediate step)
double *tempdiagonal; //for storing diagonal elements of tempWmatrix: NOTE: CAN WE REUSE temp1dvec INSTEAD??? CHECK PLEASE

// K is the covariance matrix (will be updated depending upon hyper params
double *K;
double *a11;
double *a21_transpose;
double *l21_transpose_from_fs;
double *l21;
double *l22_temp;  //This is for updating a22


double *X; // training set
double *labels; // labels of the training set (actually regression values)
double *temp_fs; // for saving the result of forward substitution while performing compute_likelihood!!
double *temp_bs; // for saving the result of backward substitution in compute_likelihood
double *ll_dotprod; // for saving the result of the dot product in compute_likelihood
 
double *loghyper;
double *log_det; // log of determinant
double *identity; // for gradient of hp

// N is the number of training samples, and DIM is the number of parameters
int N, DIM;
int totalN; //total number of samples in the dataset; totalN = N + Ntest;


int Ntest; //Ntest is the number of test samples
double *Xtest;
double *labelstest;
double *tmeanvec;
double *tvarvec;
double *Ktest_vec;

// For actual TMI
double *tmi_intermediate_output;

// For testing TMI
 double *lower_triangular_mat;
double *tmi_playground; 

#define cudacall(call) \
{ \
	hipError_t err = (call);                                                                                               \
	if(hipSuccess != err)                                                                                                  \
	{                                                                                                                       \
		fprintf(stderr,"CUDA Error:\nFile = %s\nLine = %d\nReason = %s\n", __FILE__, __LINE__, hipGetErrorString(err));    \
		hipDeviceReset();                                                                                                  \
		exit(EXIT_FAILURE);                                                                                                 \
	}                                                                                                                       \
} \

double *get_loghyperparam();


__global__ void generate_identity(double *M, int size){
	int i_index = (blockIdx.x * blockDim.x + threadIdx.x);
	int j_index = (blockIdx.y * blockDim.y + threadIdx.y);
	
	//if (i_index >= size * size)
	//		return;
//	int i = i_index / size;
//	int j = i_index % size;
//	if(i == j) M[i*size + i] = 1.0;
//	else M[i*size + j] = 0.0;	

	if(i_index >= size || j_index >= size) return;
	int mainpoint = j_index * size + i_index;
	if(i_index == j_index) M[mainpoint] = 1.0;
	else M[mainpoint] = 0.0;
}
__global__ void print_vector(double *input, int size){
	int i_index = (blockIdx.x * blockDim.x + threadIdx.x);
        if(i_index >= 1) return;
	for(int i = 0; i < size; i++){
		printf("%lf ", input[i]);
	}
	printf("\n");
}


__global__ void compute_NLPP(double *actualtestlabels, double * predicted_testmean, double *predicted_testvar, int Ntest, double * ans_nlpp){
	int i_index = (blockIdx.x * blockDim.x + threadIdx.x);
        if(i_index >= 1) return;
	double ans = 0.0;
        for(int i = 0; i < Ntest; i++) {
                double val = 0.5 * log(6.283185 * predicted_testvar[i]) + pow( (predicted_testmean[i] - actualtestlabels[i]) , 2) / (2 * predicted_testvar[i]);
		printf("predvar = %lf, predmean = %lf, actualmean = %lf, lpp = %lf\n", predicted_testvar[i], predicted_testmean[i], actualtestlabels[i], val);
                ans += val;
        }
	printf("TO FINAL ANSWER YEH HONA CHAHHIYE: %lf\n", ans / Ntest);
        *ans_nlpp = (ans / Ntest);

}

__global__ void copy_Kmatrix(double *input, double *output, int size){
	int i_index = (blockIdx.x * blockDim.x + threadIdx.x);
	int j_index = (blockIdx.y * blockDim.y + threadIdx.y);
	
	/*(if (i_index >= size * size)
		return;
	output[i_index] = input[i_index];*/
	if(i_index >= size || j_index >= size) return;
	output[j_index * size + i_index] = input[j_index * size + i_index];

}

__global__ void gather_diagonal(double *inputMat, double *diagvec, int size){
	
	int i_index = (blockIdx.x * blockDim.x + threadIdx.x);
	int j_index = (blockIdx.y * blockDim.y + threadIdx.y);
	if (i_index >= size)
		return;
	diagvec[i_index] = inputMat[i_index * size + i_index];

}

__global__ void set_upper_zero(double *M, int dim){

	int i_index = (blockIdx.x * blockDim.x + threadIdx.x);
	int j_index = (blockIdx.y * blockDim.y + threadIdx.y);

/*	if (i_index >= (dim * dim))
		return;

	int rowN = i_index / dim;
	int colN = i_index % dim;

	if(rowN >= colN) return;

	M[rowN * dim + colN] = 0.0;*/


	if(i_index >= dim || j_index >= dim || j_index >= i_index) return;
	M[j_index*dim + i_index] = 0.0;	
}

	__global__ void
get_determinant_from_L(double *M, int dim, double *log_det)
{
	// single thread

	double ans = 0.0;
	for (int i = 0; i < dim; i++){
		double val = log(M[i * dim + i]);
		ans += val;
	}
	ans *= 2;
	*log_det = ans;
	printf("Determinant is %lf\n", ans);
}

	__global__ void
elementwise_matrix_mult(double *mat1, double *mat2, double *mat3, int rows, int cols)
{
	int i_index = (blockIdx.x * blockDim.x + threadIdx.x);
	int j_index = (blockIdx.y * blockDim.y + threadIdx.y);
	
	/*
	int target_row, target_col;
	double dot_product = 0.0;

	target_row = i_index / cols;
	target_col = i_index % cols;

	if (target_row >= rows || target_col >= cols)
		return;

	mat3[target_row * cols + target_col] = mat1[target_row * cols + target_col] * mat2[target_row * cols + target_col];
	*/

	if(i_index >= cols || j_index >= rows) return;
	mat3[j_index * rows + i_index ] = mat1[j_index * rows + i_index] * mat2[j_index * rows + i_index];
}

__global__ void
compute_K_train(double *M, double *K_output, double *loghyper, int n, int dim) {
	int i_index = (blockIdx.x * blockDim.x + threadIdx.x);
	int j_index = (blockIdx.y * blockDim.y + threadIdx.y);

/*	if (i_index >= n * n) return;

	double ell_sq = exp(loghyper[0] * 2); //l^2 after coverting back from the log form
	double signal_var = exp(loghyper[1] * 2); // signal variance
	double noise_var = exp(loghyper[2] * 2); //noise variance

	int M_row, M_col;

	M_row = i_index / n;
	M_col = i_index % n;

	if (M_row < M_col) // upper triangular bye bye
		return;

	if (M_row == M_col){
		K_output[M_row * n + M_col] = signal_var +  noise_var;
		return;
	}

	for (int i = 0; i < dim; i++)
		dot_product += (M[M_row * dim + i] - M[M_col * dim + i]) * (M[M_row * dim + i] - M[M_col * dim + i]);

	dot_product = signal_var * exp(-dot_product * 0.5 / ell_sq);

	K_output[M_row * n + M_col] = K_output[M_col * n + M_row] = dot_product;
*/
	if (i_index >= n || j_index >= n || i_index > j_index) return;

	double ell_sq = exp(loghyper[0] * 2); //l^2 after coverting back from the log form
	double signal_var = exp(loghyper[1] * 2); // signal variance
	double noise_var = exp(loghyper[2] * 2); //noise variance

	double dot_product = 0.0;
	if(i_index == j_index){
		K_output[j_index * n + i_index] = signal_var + noise_var;
		return;
	}
	for(int i = 0 ; i < dim;i++){
		double val1 = M[i_index * dim + i];
		double val2 = M[j_index * dim + i];
		dot_product += (val1 - val2) * (val1 - val2);
	}
	dot_product = signal_var * exp(-dot_product * 0.5 / ell_sq);
	K_output[j_index * n + i_index] = K_output[i_index * n + j_index] = dot_product;
}

__global__ void
compute_K_test(double *M, double *testsample, double *ktest_output_vector, double *loghyper, int n, int dim) {
	int i_index = (blockIdx.x * blockDim.x + threadIdx.x);
	int j_index = (blockIdx.y * blockDim.y + threadIdx.y);

	if (i_index >= n ) return;

	double ell_sq = exp(loghyper[0] * 2); //l^2 after coverting back from the log form
	double signal_var = exp(loghyper[1] * 2); // signal variance
	double noise_var = exp(loghyper[2] * 2); //noise variance

	double dot_product = 0.0;


	for (int i = 0; i < dim; i++){
		double val1 = M[i_index * dim + i];
		double val2 = testsample[i];
		dot_product += (val1 - val2) * (val1 - val2);
	}

	dot_product = signal_var * exp(-dot_product * 0.5 / ell_sq);

	ktest_output_vector[i_index] = dot_product;
}

__global__ void matrix_vector_multiply(double *M, double *x, double *output, int size) {
	
	int i_index = (blockIdx.x * blockDim.x + threadIdx.x);
	int j_index = (blockIdx.y * blockDim.y + threadIdx.y);

	if (i_index >= size) return;
	
	double ans = 0.0;
	for(int i = 0; i < size; i++){
		ans += M[i_index * size + i] * x[i];
	}
	output[i_index] = ans;
}

__global__ void vector_matrix_multiply(double *x, double *M, double *output, int size) {
	
	int i_index = (blockIdx.x * blockDim.x + threadIdx.x);
	int j_index = (blockIdx.y * blockDim.y + threadIdx.y);

	if (i_index >= size) return;
	
	double ans = 0.0;
	for(int i = 0; i < size; i++){
		ans += M[i * size + i_index] * x[i];
	}
	output[i_index] = ans;
}

__global__ void
compute_squared_distances(double *M, double *compute_squared_distances_matrix, double *loghyper, int n, int dim) {
	int i_index = (blockIdx.x * blockDim.x + threadIdx.x);
	int j_index = (blockIdx.y * blockDim.y + threadIdx.y);

/*	if(i_index >= n * n) return;

	int M_row, M_col;
	double ell_sq = exp(loghyper[0] * 2); //l^2 after coverting back from the log form

	M_row = i_index / n;
	M_col = i_index % n;

	if (M_row < M_col) // upper triangular bye bye
		return;

	if (M_row == M_col)
	{
		compute_squared_distances_matrix[M_row * n + M_col] = 0.0;
		return;
	}
	double dot_product = 0.0;
	for (int i = 0; i < dim; i++)
		dot_product += (M[M_row * dim + i] - M[M_col * dim + i]) * (M[M_row * dim + i] - M[M_col * dim + i]);

	compute_squared_distances_matrix[M_row * n + M_col] = compute_squared_distances_matrix[M_col * n + M_row] = dot_product / ell_sq;

*/
	if(i_index >= n || j_index >= n || i_index > j_index) return;
	double ell_sq = exp(loghyper[0] * 2); //l^2 after coverting back from the log form
	if(i_index == j_index){
		compute_squared_distances_matrix[j_index * n + i_index] = 0.0;
		return;
	}	
	double dot_product = 0.0;
	for (int i = 0; i < dim; i++){
		double val1 = M[i_index * dim + i];
		double val2 = M[j_index * dim + i];
		dot_product += (val1 - val2) * (val1 - val2);
	}
	dot_product /= ell_sq;
	compute_squared_distances_matrix[i_index * n + j_index] = dot_product;
	compute_squared_distances_matrix[j_index * n + i_index] = dot_product;
}

__inline__ int upit(int x, int y) {
	return (x + y - 1) / y;
}


void init_and_print()
{
	int deviceCount = 0;
	bool isFastGPU = false;
	std::string name;
	hipError_t err = hipGetDeviceCount(&deviceCount);

	printf("---------------------------------------------------------\n");
	printf("Initializing CUDA for cuGP\n");
	printf("Found %d CUDA devices\n", deviceCount);

	for (int i=0; i<deviceCount; i++) {
		hipDeviceProp_t deviceProps;
		hipGetDeviceProperties(&deviceProps, i);
		name = deviceProps.name;
		if (name.compare("GeForce GTX 480") == 0
				|| name.compare("GeForce GTX 670") == 0
				|| name.compare("GeForce GTX 780") == 0)
		{
			isFastGPU = true;
		}

		printf("Device %d: %s\n", i, deviceProps.name);
		printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
		printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
		printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
	}
	printf("---------------------------------------------------------\n");
	if (!isFastGPU)
	{
		printf("WARNING: "
				"You're not running on a fast GPU, please consider using "
				"NVIDIA GTX 480, 670 or 780.\n");
		printf("---------------------------------------------------------\n");
	}
}

void read_input_and_copy_to_GPU(int numtrain, std::string inputfilename, std::string outputfilename)
{
	printf("Inside read_input_and_copy_to_GPUs\n");
	FILE *input_file, *label_file;
	double *X_host; //input dataset in host!
	double *labels_host; //labels in host!
	lh_host = new double[3];

	input_file = fopen(inputfilename.c_str(), "r");
	label_file = fopen(outputfilename.c_str(), "r");

	fscanf(input_file, "%d%d", &totalN, &DIM);

	for (int i = 0 ; i < 3 ; i++)
		lh_host[i] = 0.5;	

	
	X_host = new double[totalN * DIM];
	labels_host = new double[totalN];

	N = numtrain;

	printf("Reading inputs boy\n");
	printf("Number of inputs = %d\n", totalN);
 
	// Reading inputs
	for (int i = 0; i < totalN; i++)
		for (int j = 0; j < DIM; j++)
			fscanf(input_file, "%lf", &X_host[i * DIM + j]);

	// Reading labels (target values)
	for (int i = 0; i < totalN; i++) {
                fscanf(label_file, "%lf", &labels_host[i]);
        }
	
	printf("reading labels \n");
	cudacall(hipMalloc(&X, sizeof(double) * totalN * DIM));
	cudacall(hipMemcpy(X, X_host, sizeof(double) * totalN * DIM, hipMemcpyHostToDevice));	

	cudacall(hipMalloc(&labels, sizeof(double) * totalN ));
	cudacall(hipMemcpy(labels, labels_host, sizeof(double) * totalN , hipMemcpyHostToDevice));	
	
	cudacall(hipMalloc(&loghyper, sizeof(double) * 3));
	cudacall(hipMemcpy(loghyper, lh_host, sizeof(double) * 3 , hipMemcpyHostToDevice));	
	
	printf("Okay boy.. reading and malloc done\n\n");
}

void setup_loglikelihood_data()
{
	// this is the covariance matrix
	cudacall(hipMalloc(&K, sizeof(double) * N * N));
	
	//matrix for storing K.inverse()
	cudacall(hipMalloc(&Kinv, sizeof(double) * N * N));

	//matrix for storing K.inverse()
	cudacall(hipMalloc(&lowermat_inv_store, sizeof(double) * N * N));
	
	// this is the log determinant
	cudacall(hipMalloc(&log_det, sizeof(double)));

	// this is for the dot product
	cudacall(hipMalloc(&ll_dotprod, sizeof(double)));
	
	cudacall(hipMalloc(&temp_fs, sizeof(double) * N));
	cudacall(hipMalloc(&temp_bs, sizeof(double) * N));
		
	// just for checking cholesky correctness, delete later FIXME
	orig_sym = new double[N * N]; // should be equal to covariance matrix
}

void setup_cholesky(int, int);

void setup_gradienthp_data(){
	int threads_per_block, number_of_blocks;
	
	// this is the squarred distance matrix
	cudacall(hipMalloc(&Ksqdist, sizeof(double) * N * N));
	
	cudacall(hipMalloc(&matforell, sizeof(double) * N * N));

	// CHECK: if we can get away without this
	cudacall(hipMalloc(&identity, sizeof(double) * N *N));
	cudacall(hipMemset((void *)identity, 0.0, sizeof(double) * N * N));

	//matrix for storing forward substitution result
	cudacall(hipMalloc(&tempfsforkinv, sizeof(double) * N *N));
		
	
	// Kintact = K (From compute_K_train)
	cudacall(hipMalloc(&Kintact, sizeof(double) * N * N));
	
	cudacall(hipMalloc(&temp1dvec, sizeof(double) * N));
	
	cudacall(hipMalloc(&tempdiagonal, sizeof(double) * N));

	cudacall(hipMalloc(&tempWmatrix, sizeof(double) * N * N));

	
}

void setup_TMI()
{
	cudacall(hipMalloc(&tmi_intermediate_output, sizeof(double) * N * N));
}

void destruct_cublas_cusoler(){
	hipsolverDnDestroy(solver_handle);
	hipblasDestroy ( blas_handle );
}
void setup_cublas_cusolver(){
		
        cudacall(hipMalloc(&devInfo, sizeof(int)));
        hipsolverDnCreate(&solver_handle);
	stat = hipblasCreate (& blas_handle );
}
void setup( int numtrain, std::string inputfilename, std::string outputfilename)
{
	printf("YEEEEEEEEEEEEEEEHH setup call huaa\n");
	std::string s;
	read_input_and_copy_to_GPU(numtrain, inputfilename, outputfilename);

	setup_loglikelihood_data();

	setup_cholesky(N, 2);
	
	setup_gradienthp_data();	

	setup_TMI();
	
	setup_cublas_cusolver();
}

void setup_cholesky(int dim, int b)
{
	/* cudacall(hipMalloc(&mt, sizeof(double) * dim * dim));
	   cudacall(hipMalloc(&mt_transpose, sizeof(double) * dim * dim));
	   cudacall(hipMemcpy(mt, temp_m, sizeof(double) * dim * dim, hipMemcpyHostToDevice)); */

	/*
	 * Now malloc the a11 matrix
	 */

	cudacall(hipMalloc(&a11, sizeof(double) * 4));

	/*
	 * Now malloc the a21_transpose matrix by overprovisioning. This can be of maximum size bx(dim-b). But, we allocate
	 * a bx(dim-b) vector even for the latter stages.
	 */

	cudacall(hipMalloc(&a21_transpose, sizeof(double) * b * (dim - b)));
	cudacall(hipMemset((void *)a21_transpose, 0, sizeof(double) * b * (dim - b)));

	/*
	 * Now malloc the l21_transpose_from_fs matrix to insert the output of forward substitution. Is retained here for generating a22.
	 */

	cudacall(hipMalloc(&l21_transpose_from_fs, sizeof(double) * b * (dim - b)));
	cudacall(hipMemset((void *)l21_transpose_from_fs, 0, sizeof(double) * b * (dim - b)));

	/*
	 * Now malloc the l21 matrix, which will be useful for populating a22 (via matrix mult).
	 */

	cudacall(hipMalloc(&l21, sizeof(double) * b * (dim - b)));

	/*
	 * Now malloc the l22_temp matrix, which will be useful for elementwise subtraction for a22 (after matrix mult).
	 */

	cudacall(hipMalloc(&l22_temp, sizeof(double) * (dim - b) * (dim - b)));

}



// M is a device pointer!!, n is the size of the matrix
void get_cholesky_using_cublas(double *M, int n){

	printf("Haaaar REE call hua\n");
	double startime = CycleTimer::currentSeconds();	
        int work_size = 0;


	//print_matrix_kernel<<<1,1>>>(M, n, n);
	//hipDeviceSynchronize(); 
        
	// --- CUDA CHOLESKY initialization: Not needed
         hipsolverDnDpotrf_bufferSize(solver_handle, HIPBLAS_FILL_MODE_UPPER, n, M, n * n, &work_size);
        
	// --- CUDA POTRF execution
	 double *work;
  	 cudacall(hipMalloc(&work, work_size * sizeof(double)));
   	 hipsolverDnDpotrf(solver_handle, HIPBLAS_FILL_MODE_UPPER, n, M, n, work, work_size, devInfo);

	// Giving the entire Kinv for buffer: Be happy!!
        //hipsolverDnDpotrf(solver_handle, HIPBLAS_FILL_MODE_UPPER, n, M, n, Kinv, n * n, devInfo);
	
       int devInfo_h = 0;
       cudacall(hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost));
        if (devInfo_h != 0) std::cout   << "Unsuccessful potrf execution, value = " << devInfo_h << std::endl;
	else printf("okay cholesky went fine\n");

	double endtime = CycleTimer::currentSeconds();
	printf("Time taken for CUSOLVER cholesky: %lf\n", endtime - startime);
	//now make upper vala 0
/*	int threads_per_block = 1024;
        int number_of_blocks = upit( (n * n), threads_per_block);
        set_upper_zero<<<number_of_blocks, threads_per_block>>>(M, n);
        hipDeviceSynchronize();
*/

	dim3 blockDim(32,32);
 	dim3 gridDim( upit(n, blockDim.x), upit(n, blockDim.y));
        set_upper_zero<<<gridDim, blockDim>>>(M, n);
        hipDeviceSynchronize();

}


void get_inverse_by_cublas(double *Lmat, int sizelmat){
        double al =1.0f;

	int threads_per_block, number_of_blocks;
	//threads_per_block = 1024;
	//number_of_blocks = upit((sizelmat * sizelmat), threads_per_block);
	//generate_identity<<<number_of_blocks, threads_per_block>>>(lowermat_inv_store, sizelmat);

	dim3 blockDim(32,32);
 	dim3 gridDim( upit(N, blockDim.x), upit(N, blockDim.y));
	generate_identity<<<gridDim, blockDim>>>(lowermat_inv_store, sizelmat);
	hipDeviceSynchronize(); 
		
        double startime = CycleTimer::currentSeconds();
        (hipblasDtrsm(blas_handle,HIPBLAS_SIDE_LEFT,HIPBLAS_FILL_MODE_UPPER,
                        HIPBLAS_OP_N,HIPBLAS_DIAG_NON_UNIT, sizelmat , sizelmat ,&al, Lmat, sizelmat, lowermat_inv_store, sizelmat));
        double endtime = CycleTimer::currentSeconds();
	printf("time taken by cublas-TMI-inverse = %lf\n", endtime - startime);
}

void matrix_multiply_cublas_withtranspose(double *A, double *B, double *C, int size){
     const double alf = 1;
     const double bet = 0;
     const double *alpha = &alf;
     const double *beta = &bet;

     // Do the actual multiplication
     hipblasDgemm(blas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, size, size, size, alpha, A, size, B, size, beta, C, size);
}

void matrix_vector_multiply_cublas(double *A, double *B, double *C, int size){
     const double alf = 1;
     const double bet = 0;
     const double *alpha = &alf;
     const double *beta = &bet;

     // Do the actual multiplication
     hipblasDgemv(blas_handle, HIPBLAS_OP_N, size, size, alpha, A, size, B, 1, beta, C, 1);
}



void get_inverse_by_tmi(double *, int );
void compute_chol_get_mul_and_det()
{
	int threads_per_block, number_of_blocks;


//	get_cholesky(K, N); // set of kernels : Now K is actually L
	get_cholesky_using_cublas(K, N); // set of kernels

//	printf("INVERSE KE BAAD\n");
//	print_matrix_kernel<<<1,1>>>(K, N, N);
//	hipDeviceSynchronize(); 

	get_determinant_from_L<<<1, 1>>>(K, N, log_det);
	hipDeviceSynchronize();
	
 	/* // forward_solve_vector(); // kernel K * y = target -> solves for y (Note K is a lower triangular matrix)
	threads_per_block = 512;
	number_of_blocks = upit(N, threads_per_block);
	forward_substitution_vector<<<1, 1>>>(K, labels, temp_fs, N);
	hipDeviceSynchronize();

	// backward_solve_vector(); // kernel L_transpose * x = y -> solves for x 
	backward_substitution_vector<<<1, 1>>>(K, temp_fs, temp_bs, N); // Since we use the K.transpose() inside we don't pass L_transpose
	hipDeviceSynchronize();
	
	// compute_product(); // kernel
	vector_dot_product<<<1, 1>>>(temp_bs, labels, ll_dotprod, N);
	hipDeviceSynchronize();

	//LMI of K
	//vector-matrix-mulitply of (ouput of prev step) and labels => temp_bs
	//TODO: CAN DELETE TEMP_FS..... */

	printf("okay NOW TMI with cublas instead of our call\n");
//	get_inverse_by_tmi(K, N);
//     	hipDeviceSynchronize();
	get_inverse_by_cublas(K, N); //-> Now note that the result will be in lowermat_inv_store and not K (IMP)


//      NOTE: THIS TRANPOSE IS NOT REQUIRED
//		BECAUSE WE HAVE AN ADHOC MATRIX_MULITPLY_CUBLAS_WITHTRANPOSE
//	double alpha = 1.0, beta = 0.0;
//	//now we need to get the transpose lowermat_inv_store : let's store this in K (reuse!!!)
//	hipblasDgeam(blas_handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, N, &alpha, lowermat_inv_store, N, &beta, lowermat_inv_store, N, K, N);	

//	printf("Bhai transpose dekhle \n");
//	print_matrix_kernel<<<1, 1>>>(K,N, N);
//	hipDeviceSynchronize(); 

	//Now we will do the DGEMM matrix multiply	
//	printf("SIMPLE MULTIPLY\n");
//	dim3 blockDim(32,32);
 //       dim3 gridDim( upit(N, blockDim.x), upit(N, blockDim.y));
  //      lowertriangular_matrixmultiply_noshare<<<gridDim, blockDim >>>(lowermat_inv_store, Kinv, N);
  //      hipDeviceSynchronize();
	
	matrix_multiply_cublas_withtranspose(lowermat_inv_store, lowermat_inv_store, Kinv, N);
	//printf("Bhai DGEMM dekhle \n");
	//print_matrix_kernel<<<1, 1>>>(Kinv,N, N);
	//hipDeviceSynchronize(); 
	
	

	matrix_vector_multiply_cublas(Kinv, labels, temp_bs, N);
	//printf("\nokay neeche dekho\n");
	//print_vector<<<1, 1>>>(temp_bs, N);
	//hipDeviceSynchronize(); 

	/*	
        threads_per_block = 512;
        number_of_blocks = upit(N, threads_per_block);
        matrix_vector_multiply<<<number_of_blocks, threads_per_block>>>(Kinv, labels, temp_bs, N);
        hipDeviceSynchronize();

	printf("\n\nokay DOT PRODUCT YEH AYA HAI\n");
	print_vector<<<1, 1>>>(temp_bs, N);
	hipDeviceSynchronize(); 
	
	vector_dot_product<<<1, 1>>>(temp_bs, labels, ll_dotprod, N);
	hipDeviceSynchronize();
	*/

	

	thrust::device_ptr<double> td1 = thrust::device_pointer_cast(temp_bs);
	thrust::device_ptr<double> td2 = thrust::device_pointer_cast(labels);

	double ans = 0.0;
	ans = thrust::inner_product(td1, td1 + N, td2, 0.0);
	printf("\n\n ############# PRODUCT YEH AYA = %lf\n", ans);
	//POTENTIAL IMPROVEMENT POSSIBLE, can return ans; instead of doing transfer shittt, 
	cudacall(hipMemcpy(ll_dotprod, &ans, sizeof(double), hipMemcpyHostToDevice));

}


double evaluate_and_get_log_likelihood(){
	double term1_ll;
	double term2_ll;
	cudacall(hipMemcpy(&term1_ll, ll_dotprod,  sizeof(double), hipMemcpyDeviceToHost));
	cudacall(hipMemcpy(&term2_ll, log_det ,  sizeof(double), hipMemcpyDeviceToHost));
	printf("product vala term = %lf\n", term1_ll);
	return -0.5 * ( term1_ll + term2_ll + N * 1.83787);
}
double compute_log_likelihood()
{
	int threads_per_block, number_of_blocks;

	double compute_ll_start = CycleTimer::currentSeconds();

	printf("compute_K_train hota hai\n");

	dim3 blockDim1(32,32);
 	dim3 gridDim1( upit(N, blockDim1.x), upit(N, blockDim1.y));
	double ckts = CycleTimer::currentSeconds();
	compute_K_train<<<gridDim1, blockDim1>>>(X, K, loghyper, N, DIM); // kernel
	hipDeviceSynchronize();
	double ckte = CycleTimer::currentSeconds();
	printf("compute_K_train time: %lf\n", ckte - ckts);

	compute_chol_get_mul_and_det(); // set of kernels

	double llans = evaluate_and_get_log_likelihood(); // kernel, or can be clubbed somewhere
	printf("The value of loglikelihood = %lf\n", llans);
	double compute_ll_end = CycleTimer::currentSeconds();
	printf("compute_log_likelihood Time: %lf\n", compute_ll_end - compute_ll_start);
	return llans; 
}

void compute_K_inverse_using_cublas()
{
	int threads_per_block, number_of_blocks;
	
	// make_identity(); -> did this in setup "identity" is a double *
	double st = CycleTimer::currentSeconds();

	// SUB: get_cholesky(K, N); //Set of kernels, the answer (a lower triangular matrix) is stored 
	double chol_st = CycleTimer::currentSeconds();
	get_cholesky_using_cublas(K, N); // NOW result is in K
	double chol_end = CycleTimer::currentSeconds();
	printf("get_cholesky_using_cublas time: %lf\n", chol_end - chol_st);

	/* SUB:
	threads_per_block = 512;
	number_of_blocks = upit(N, threads_per_block);
	forward_substitution_matrix<<<number_of_blocks, threads_per_block>>>(K, identity, tempfsforkinv, N); // kernel - need N threads
	hipDeviceSynchronize();
	
	// matrix_transpose(); // kernel - Not NEEDED

	// matrix_backward_substitution();
	backward_substitution_matrix<<<number_of_blocks, threads_per_block>>>(K, tempfsforkinv, Kinv, N); // kernel - need N threads
	hipDeviceSynchronize();
	*/

	get_inverse_by_cublas(K, N); //-> Now note that the result will be in lowermat_inv_store and not K (IMP)

	matrix_multiply_cublas_withtranspose(lowermat_inv_store, lowermat_inv_store, Kinv, N);
	double end = CycleTimer::currentSeconds();
	printf("compute_K_inverse_using_cublas time: %lf\n", end - st);
	//-> So the answer is in Kinv
}

void compute_gradient_log_hyperparams(double *localhp_grad)
{
	double start = CycleTimer::currentSeconds();
	int threads_per_block, number_of_blocks;

	double *tt = get_loghyperparam(); //just for a MEMCPY from device to host
	double noise_var = exp(lh_host[2] * 2); //noise variance


	dim3 blockDim1(32,32);
 	dim3 gridDim1( upit(N, blockDim1.x), upit(N, blockDim1.y));
        compute_K_train<<<gridDim1, blockDim1>>>(X, K, loghyper, N, DIM); // kernel
        hipDeviceSynchronize();

	dim3 blockDim2(32,32);
 	dim3 gridDim2( upit(N, blockDim2.x), upit(N, blockDim2.y));
	copy_Kmatrix<<<gridDim2, blockDim2>>>(K, Kintact, N);
        hipDeviceSynchronize();
	
	dim3 blockDim3(32,32);
 	dim3 gridDim3( upit(N, blockDim3.x), upit(N, blockDim3.y));
   	compute_squared_distances<<<gridDim3, blockDim3>>>(X,  Ksqdist,  loghyper,  N, DIM);
   	hipDeviceSynchronize();

	dim3 blockDim4(32,32);
 	dim3 gridDim4( upit(N, blockDim4.x), upit(N, blockDim4.y));
	elementwise_matrix_mult<<<gridDim4, blockDim4>>>(K, Ksqdist, matforell, N, N);
	hipDeviceSynchronize();

	compute_K_inverse_using_cublas(); // set of kernels

	matrix_vector_multiply_cublas(Kinv, labels, temp1dvec, N); //can use temp_bs only

	dim3 blockDim5(32,32);
 	dim3 gridDim5( upit(N, blockDim5.x), upit(N, blockDim5.y));
	copy_Kmatrix<<<gridDim5, blockDim5>>>(Kinv, tempWmatrix, N);
        hipDeviceSynchronize();


	const double alf = -1; //NOTE: we neeed to subtract, that's why -1
     	const double *alpha = &alf;

	hipblasDger(blas_handle, N, N, alpha,  temp1dvec, 1, temp1dvec, 1, tempWmatrix, N);

        threads_per_block = 1024;
        number_of_blocks = upit( N, threads_per_block);
	gather_diagonal<<<number_of_blocks, threads_per_block>>>(tempWmatrix, tempdiagonal, N);	
      	hipDeviceSynchronize();
	
	// Now update_log_hyperparams(); 
	thrust::device_ptr<double> td1 = thrust::device_pointer_cast(tempWmatrix);
	thrust::device_ptr<double> td2 = thrust::device_pointer_cast(matforell);
	thrust::device_ptr<double> td3 = thrust::device_pointer_cast(Kintact);

	thrust::device_ptr<double> td4 = thrust::device_pointer_cast(tempdiagonal);

	double para1 = 0.0, para2 = 0.0, para3 = 0.0;

	para1 = thrust::inner_product(td1, td1 + N*N, td2, 0.0);
	para2 = thrust::inner_product(td1, td1 + N*N, td3, 0.0);
	para2 = para2 * 2;
	printf("Why yaar? para2 = %lf\n", para2);
	double common_val = thrust::reduce(td4, td4 + N);
	common_val *= noise_var * 2;

	para3 = common_val;
	para2 = para2 - common_val;

	localhp_grad[0] = para1/2.0;
	localhp_grad[1] = para2/2.0;
	localhp_grad[2] = para3/2.0;
	printf("Dekho bhai %lf\n%lf\n%lf\n", localhp_grad[0], localhp_grad[1], localhp_grad[2]);
	double end = CycleTimer::currentSeconds();
	printf("compute_gradient_hyperparam time: %lf\n", end - start);

}


double *get_loghyperparam(){
	cudacall(hipMemcpy(lh_host, loghyper,  sizeof(double) * 3, hipMemcpyDeviceToHost));	
	return  lh_host;
}

void set_loghyper_eigen(Eigen::VectorXd initval) {
        for(int i = 0 ; i < 3; i++) { 
                lh_host[i] = initval[i];
        }
	printf("Dekho bhai naya value AYAA\n\n");
	for(int i = 0 ; i < 3 ; i++){
		printf("%lf\n", lh_host[i]);
	}
	//Now dump it back to loghyper
	cudacall(hipMemcpy(loghyper, lh_host, sizeof(double) * 3 , hipMemcpyHostToDevice)); 
}


void setup_for_testing(int offset, int numtest){
	Xtest = X + DIM * offset;
	labelstest = labels + offset;
	Ntest = numtest;	
	
	cudacall(hipMalloc(&tmeanvec, sizeof(double) * Ntest));
	cudacall(hipMalloc(&tvarvec, sizeof(double) * Ntest));
	
	//Remember Ktest_vec should have size = N, not Ntest
	cudacall(hipMalloc(&Ktest_vec, sizeof(double) * N));

}

//compute_test_means_and_variances is a set of kernels: FIX testing code
void compute_test_means_and_variances(){
	int threads_per_block, number_of_blocks;

	//Maybe can move the compute_K_train to setup in SCHEDULER-vala (THINK ABOUT IT SID)
        threads_per_block = 1024;
        number_of_blocks = upit((N * N), threads_per_block);
    	// FIXME
	//    compute_K_train<<<number_of_blocks, threads_per_block>>>(X, K, loghyper, N, DIM); // populated in K
        hipDeviceSynchronize();
	
	//compute_K_inverse(); //populates Kinv with K.inverse()
	// instead of compute_K_inverse, let's see if TMI is of help!!!
	
	//FIXME: cublas vala call KAROOOOOOOOOOOOOOO
	//compute_K_inverse_with_tmi();	
	
	// vector_Kinvy_using_cholesky(); // set of kernels
	// We don't need this: we already have Kinv, so we just need to multiply Kinv and y
        threads_per_block = 512;
        number_of_blocks = upit( N, threads_per_block);
	matrix_vector_multiply<<<number_of_blocks, threads_per_block>>>(Kinv, labels, temp1dvec, N); //so temp1dvec gets populated
      	hipDeviceSynchronize();

	
	double sig_var = exp(lh_host[1] * 2); //signal variance
	double noise_var = exp(lh_host[2] * 2); //noise variance
	for(int i = 0; i < Ntest; i++){

        	threads_per_block = 512;
	        number_of_blocks = upit( N, threads_per_block);
		compute_K_test<<<number_of_blocks, threads_per_block>>>(X, Xtest + i * DIM, Ktest_vec, loghyper, N, DIM); //REUSE SOME ALREADY EXISTING MALLOC'ED 1D VECTOR
      		hipDeviceSynchronize();

		//FIXME
		//vector_dot_product<<<1, 1>>>(Ktest_vec, temp1dvec, tmeanvec + i, N); //for mean
      		//hipDeviceSynchronize();
	
		//FIXME
		//threads_per_block = 512;
	       // number_of_blocks = upit(N, threads_per_block);
       		//vector_matrix_multiply<<<number_of_blocks, threads_per_block>>>(Ktest_vec, Kinv, temp_fs, N); //REUSING temp_fs from likelihood computation
        //	hipDeviceSynchronize();
		//FIXME		
//		vector_dot_product_with_loghp<<<1, 1>>>(Ktest_vec, temp_fs, tvarvec + i, N, sig_var, noise_var ); //for variance
      //		hipDeviceSynchronize();
		
	}
}

void get_negative_log_predprob(){
	
	double *finalans = new double; //for host
	double *ans_nlpp;
	cudacall(hipMalloc(&ans_nlpp, sizeof(double) ));
	
	compute_NLPP<<<1, 1>>>(labelstest, tmeanvec, tvarvec, Ntest, ans_nlpp);
      	hipDeviceSynchronize();
	
	cudacall(hipMemcpy(finalans, ans_nlpp,  sizeof(double), hipMemcpyDeviceToHost));
	printf("OKAY FINAL NLPP = %lf\n", *finalans);
}

void testing_phase(int offset, int numtest){

	printf("---------------------------------------\n");
	printf("TRYING TO START TESTING PHASE\n");	
	printf("---------------------------------------\n");
	//setup THINGS
	setup_for_testing(offset, numtest);
	
	printf("\n---------------------------------------\n");
	printf("TRYING TO Start compute_test_means PHASE\n");	
	printf("---------------------------------------\n");
	// Now calling testing phase
	double startime = CycleTimer::currentSeconds();	
	compute_test_means_and_variances();
	double endtime = CycleTimer::currentSeconds();
	printf("TESTING TIME = %lf\n", endtime - startime);
		
	printf("\n---------------------------------------\n");
	printf("Now result time\n");	
	printf("---------------------------------------\n");
	// actual answer time
	get_negative_log_predprob();
	
}
