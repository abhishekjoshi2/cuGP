#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
//#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#if __CUDA_ARCH__ >= 700
#include <hipsolver.h>
#endif
#include <hipblas.h>
#include <hipfft/hipfft.h>

#include "Utilities.cuh"

#define DEBUG

/*******************/
/* iDivUp FUNCTION */
/*******************/
extern "C" int iDivUp(int a, int b){ return ((a % b) != 0) ? (a / b + 1) : (a / b); }

/********************/
/* CUDA ERROR CHECK */
/********************/
// --- Credit to http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
	  if (abort) { exit(code); }
   }
}

extern "C" void gpuErrchk(hipError_t ans) { gpuAssert((ans), __FILE__, __LINE__); }

/**************************/
/* CUSOLVE ERROR CHECKING */
/**************************/
#if __CUDA_ARCH__ >= 700
static const char *_cusolverGetErrorEnum(hipsolverStatus_t error)
{
    switch (error)
    {
        case HIPSOLVER_STATUS_SUCCESS:
            return "CUSOLVER_SUCCESS";

        case HIPSOLVER_STATUS_NOT_INITIALIZED:
            return "HIPSOLVER_STATUS_NOT_INITIALIZED";

        case HIPSOLVER_STATUS_ALLOC_FAILED:
            return "HIPSOLVER_STATUS_ALLOC_FAILED";

        case HIPSOLVER_STATUS_INVALID_VALUE:
            return "HIPSOLVER_STATUS_INVALID_VALUE";

        case HIPSOLVER_STATUS_ARCH_MISMATCH:
            return "HIPSOLVER_STATUS_ARCH_MISMATCH";

        case HIPSOLVER_STATUS_EXECUTION_FAILED:
            return "HIPSOLVER_STATUS_EXECUTION_FAILED";

        case HIPSOLVER_STATUS_INTERNAL_ERROR:
            return "HIPSOLVER_STATUS_INTERNAL_ERROR";

        case HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
            return "HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED";

    }

    return "<unknown>";
}

inline void __cusolveSafeCall(hipsolverStatus_t err, const char *file, const int line)
{
    if(HIPSOLVER_STATUS_SUCCESS != err) {
		fprintf(stderr, "CUSOLVE error in file '%s', line %Ndims\Nobjs %s\nerror %Ndims: %s\nterminating!\Nobjs",__FILE__, __LINE__,err, \
                                _cusolverGetErrorEnum(err)); \
		hipDeviceReset(); assert(0); \
	}
}

extern "C" void cusolveSafeCall(hipsolverStatus_t err) { __cusolveSafeCall(err, __FILE__, __LINE__); }
#endif

/*************************/
/* CUBLAS ERROR CHECKING */
/*************************/
static const char *_cublasGetErrorEnum(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";

        case HIPBLAS_STATUS_NOT_SUPPORTED:
            return "HIPBLAS_STATUS_NOT_SUPPORTED";

        case HIPBLAS_STATUS_UNKNOWN:
            return "HIPBLAS_STATUS_UNKNOWN";
}

    return "<unknown>";
}

inline void __cublasSafeCall(hipblasStatus_t err, const char *file, const int line)
{
    if(HIPBLAS_STATUS_SUCCESS != err) {
		fprintf(stderr, "CUBLAS error in file '%s', line %Ndims\Nobjs %s\nerror %Ndims: %s\nterminating!\Nobjs",__FILE__, __LINE__,err, \
                                _cublasGetErrorEnum(err)); \
		hipDeviceReset(); assert(0); \
	}
}

extern "C" void cublasSafeCall(hipblasStatus_t err) { __cublasSafeCall(err, __FILE__, __LINE__); }

/************************/
/* CUFFT ERROR CHECKING */
/************************/
static const char *_cufftGetErrorEnum(hipfftResult error)
{
    switch (error)
    {
        case HIPFFT_SUCCESS:
            return "HIPFFT_SUCCESS";

        case HIPFFT_INVALID_PLAN:
            return "HIPFFT_INVALID_PLAN";

        case HIPFFT_ALLOC_FAILED:
            return "HIPFFT_ALLOC_FAILED";

        case HIPFFT_INVALID_TYPE:
            return "HIPFFT_INVALID_TYPE";

        case HIPFFT_INVALID_VALUE:
            return "HIPFFT_INVALID_VALUE";

        case HIPFFT_INTERNAL_ERROR:
            return "HIPFFT_INTERNAL_ERROR";

        case HIPFFT_EXEC_FAILED:
            return "HIPFFT_EXEC_FAILED";

        case HIPFFT_SETUP_FAILED:
            return "HIPFFT_SETUP_FAILED";

        case HIPFFT_INVALID_SIZE:
            return "HIPFFT_INVALID_SIZE";

        case HIPFFT_UNALIGNED_DATA:
            return "HIPFFT_UNALIGNED_DATA";
    }

    return "<unknown>";
}

// --- CUFFTSAFECALL
inline void __cufftSafeCall(hipfftResult err, const char *file, const int line)
{
    if( HIPFFT_SUCCESS != err) {
		fprintf(stderr, "CUFFT error in file '%s', line %d\n \nerror %d: %s\nterminating!\n",__FILE__, __LINE__,err, _cufftGetErrorEnum(err));
		hipDeviceReset(); assert(0);
    }
}

extern "C" void cufftSafeCall(hipfftResult err) { __cufftSafeCall(err, __FILE__, __LINE__); }

/***************************/
/* CUSPARSE ERROR CHECKING */
/***************************/
static const char *_cusparseGetErrorEnum(hipsparseStatus_t error)
{
    switch (error)
    {

		case HIPSPARSE_STATUS_SUCCESS:
            return "HIPSPARSE_STATUS_SUCCESS";

        case HIPSPARSE_STATUS_NOT_INITIALIZED:
            return "HIPSPARSE_STATUS_NOT_INITIALIZED";

        case HIPSPARSE_STATUS_ALLOC_FAILED:
            return "HIPSPARSE_STATUS_ALLOC_FAILED";

        case HIPSPARSE_STATUS_INVALID_VALUE:
            return "HIPSPARSE_STATUS_INVALID_VALUE";

        case HIPSPARSE_STATUS_ARCH_MISMATCH:
            return "HIPSPARSE_STATUS_ARCH_MISMATCH";

        case HIPSPARSE_STATUS_MAPPING_ERROR:
            return "HIPSPARSE_STATUS_MAPPING_ERROR";

        case HIPSPARSE_STATUS_EXECUTION_FAILED:
            return "HIPSPARSE_STATUS_EXECUTION_FAILED";

        case HIPSPARSE_STATUS_INTERNAL_ERROR:
            return "HIPSPARSE_STATUS_INTERNAL_ERROR";

        case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
            return "HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED";

        case HIPSPARSE_STATUS_ZERO_PIVOT:
            return "HIPSPARSE_STATUS_ZERO_PIVOT";
	}

    return "<unknown>";
}

inline void __cusparseSafeCall(hipsparseStatus_t err, const char *file, const int line)
{
    if(HIPSPARSE_STATUS_SUCCESS != err) {
		fprintf(stderr, "CUSPARSE error in file '%s', line %Ndims\Nobjs %s\nerror %Ndims: %s\nterminating!\Nobjs",__FILE__, __LINE__,err, \
                                _cusparseGetErrorEnum(err)); \
		hipDeviceReset(); assert(0); \
	}
}

extern "C" void cusparseSafeCall(hipsparseStatus_t err) { __cusparseSafeCall(err, __FILE__, __LINE__); }

/************************/
/* REVERSE ARRAY KERNEL */
/************************/
#define BLOCKSIZE_REVERSE	256

// --- Credit to http://www.drdobbs.com/parallel/cuda-supercomputing-for-the-masses-part/208801731?pgno=2
template <class T>
__global__ void reverseArrayKernel(const T * __restrict__ d_in, T * __restrict__ d_out, const int N, const T a)
{
	// --- Credit to the simpleTemplates CUDA sample
	SharedMemory<T> smem;
    T* s_data = smem.getPointer();

    const int tid			= blockDim.x * blockIdx.x + threadIdx.x;
	const int id			= threadIdx.x;
	const int offset		= blockDim.x * (blockIdx.x + 1);

	// --- Load one element per thread from device memory and store it *in reversed order* into shared memory
	if (tid < N) s_data[BLOCKSIZE_REVERSE - (id + 1)] = a * d_in[tid];

	// --- Block until all threads in the block have written their data to shared memory
	__syncthreads();

	// --- Write the data from shared memory in forward order
	if ((N - offset + id) >= 0) d_out[N - offset + id] = s_data[threadIdx.x];
}

/************************/
/* REVERSE ARRAY KERNEL */
/************************/
template <class T>
void reverseArray(const T * __restrict__ d_in, T * __restrict__ d_out, const int N, const T a) {

    reverseArrayKernel<<<iDivUp(N, BLOCKSIZE_REVERSE), BLOCKSIZE_REVERSE, BLOCKSIZE_REVERSE * sizeof(T)>>>(d_in, d_out, N, a);
#ifdef DEBUG
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
#endif

}

template void reverseArray<float>  (const float  * __restrict__, float  * __restrict__, const int, const float);
template void reverseArray<double> (const double * __restrict__, double * __restrict__, const int, const double);

/********************************************************/
/* CARTESIAN TO POLAR COORDINATES TRANSFORMATION KERNEL */
/********************************************************/
#define BLOCKSIZE_CART2POL	256

template <class T>
__global__ void Cartesian2PolarKernel(const T * __restrict__ d_x, const T * __restrict__ d_y, T * __restrict__ d_rho, T * __restrict__ d_theta,
	                       const int N, const T a) {

	const int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < N) {
		d_rho[tid]		= a * hypot(d_x[tid], d_y[tid]);
		d_theta[tid]	= atan2(d_y[tid], d_x[tid]);
	}

}

/*******************************************************/
/* CARTESIAN TO POLAR COORDINATES TRANSFORMATION - GPU */
/*******************************************************/
//template <class T>
//thrust::pair<T *,T *> Cartesian2Polar(const T * __restrict__ d_x, const T * __restrict__ d_y, const int N, const T a) {
//
//	T *d_rho;	gpuErrchk(hipMalloc((void**)&d_rho,   N * sizeof(T)));
//	T *d_theta; gpuErrchk(hipMalloc((void**)&d_theta, N * sizeof(T)));
//
//	Cartesian2PolarKernel<<<iDivUp(N, BLOCKSIZE_CART2POL), BLOCKSIZE_CART2POL>>>(d_x, d_y, d_rho, d_theta, N, a);
//#ifdef DEBUG
//	gpuErrchk(hipPeekAtLastError());
//	gpuErrchk(hipDeviceSynchronize());
//#endif
//
//	return thrust::make_pair(d_rho, d_theta);
//}
//
//template thrust::pair<float  *, float  *>  Cartesian2Polar<float>  (const float  *, const float  *, const int, const float);
//template thrust::pair<double *, double *>  Cartesian2Polar<double> (const double *, const double *, const int, const double);

/*******************************************************/
/* CARTESIAN TO POLAR COORDINATES TRANSFORMATION - CPU */
/*******************************************************/
//template <class T>
//thrust::pair<T *,T *> h_Cartesian2Polar(const T * __restrict__ h_x, const T * __restrict__ h_y, const int N, const T a) {
//
//	T *h_rho	= (T *)malloc(N * sizeof(T));
//	T *h_theta	= (T *)malloc(N * sizeof(T));
//
//	for (int i = 0; i < N; i++) {
//		h_rho[i]	= a * hypot(h_x[i], h_y[i]);
//		h_theta[i]	= atan2(h_y[i], h_x[i]);
//	}
//
//	return thrust::make_pair(h_rho, h_theta);
//}
//
//template thrust::pair<float  *, float  *>  h_Cartesian2Polar<float>  (const float  *, const float  *, const int, const float);
//template thrust::pair<double *, double *>  h_Cartesian2Polar<double> (const double *, const double *, const int, const double);

/*******************************/
/* LINEAR COMBINATION FUNCTION */
/*******************************/
void linearCombination(const float * __restrict__ d_coeff, const float * __restrict__ d_basis_functions_real, float * __restrict__ d_linear_combination,
	                   const int N_basis_functions, const int N_sampling_points, const hipblasHandle_t handle) {

    float alpha = 1.f;
    float beta  = 0.f;
    cublasSafeCall(hipblasSgemv(handle, HIPBLAS_OP_N, N_sampling_points, N_basis_functions, &alpha, d_basis_functions_real, N_sampling_points,
                               d_coeff, 1, &beta, d_linear_combination, 1));

}

void linearCombination(const double * __restrict__ d_coeff, const double * __restrict__ d_basis_functions_real, double * __restrict__ d_linear_combination,
	                   const int N_basis_functions, const int N_sampling_points, const hipblasHandle_t handle) {

    double alpha = 1.;
    double beta  = 0.;
    cublasSafeCall(hipblasDgemv(handle, HIPBLAS_OP_N, N_sampling_points, N_basis_functions, &alpha, d_basis_functions_real, N_sampling_points,
                               d_coeff, 1, &beta, d_linear_combination, 1));

}

/******************************/
/* ADD A CONSTANT TO A VECTOR */
/******************************/
#define BLOCKSIZE_VECTORADDCONSTANT	256

template<class T>
__global__ void vectorAddConstantKernel(T * __restrict__ d_in, const T scalar, const int N) {

	const int tid	= threadIdx.x + blockIdx.x*blockDim.x;

	if (tid < N) d_in[tid] += scalar;

}

template<class T>
void vectorAddConstant(T * __restrict__ d_in, const T scalar, const int N) {

	vectorAddConstantKernel<<<iDivUp(N, BLOCKSIZE_VECTORADDCONSTANT), BLOCKSIZE_VECTORADDCONSTANT>>>(d_in, scalar, N);

}

template void  vectorAddConstant<float> (float  * __restrict__, const float , const int);
template void  vectorAddConstant<double>(double * __restrict__, const double, const int);

/*****************************************/
/* MULTIPLY A VECTOR BY A CONSTANT - GPU */
/*****************************************/
#define BLOCKSIZE_VECTORMULCONSTANT	256

template<class T>
__global__ void vectorMulConstantKernel(T * __restrict__ d_in, const T scalar, const int N) {

	const int tid	= threadIdx.x + blockIdx.x*blockDim.x;

	if (tid < N) d_in[tid] *= scalar;

}

template<class T>
void vectorMulConstant(T * __restrict__ d_in, const T scalar, const int N) {

	vectorMulConstantKernel<<<iDivUp(N, BLOCKSIZE_VECTORMULCONSTANT), BLOCKSIZE_VECTORMULCONSTANT>>>(d_in, scalar, N);

}

template void  vectorMulConstant<float> (float  * __restrict__, const float , const int);
template void  vectorMulConstant<double>(double * __restrict__, const double, const int);

/*****************************************/
/* MULTIPLY A VECTOR BY A CONSTANT - CPU */
/*****************************************/
template<class T>
void h_vectorMulConstant(T * __restrict__ h_in, const T scalar, const int N) {

	for (int i = 0; i < N; i++) h_in[i] *= scalar;

}

template void  h_vectorMulConstant<float> (float  * __restrict__, const float , const int);
template void  h_vectorMulConstant<double>(double * __restrict__, const double, const int);

/*****************************************************/
/* FUSED MULTIPLY ADD OPERATIONS FOR HOST AND DEVICE */
/*****************************************************/
template<class T>
__host__ __device__ T fma2(T x, T y, T z) { return x * y + z; }

template float  fma2<float >(float , float , float );
template double fma2<double>(double, double, double);

/*******************/
/* MODULO FUNCTION */
/*******************/
__device__ int modulo(int val, int _mod)
{
	int P;
	if(val > 0) { (!(_mod & (_mod - 1))? P = val&(_mod-1) : P = val%(_mod)); return P; }
	else
	{
		(!(_mod & (_mod - 1))? P = (-val)&(_mod-1) : P = (-val)%(_mod));
		if(P > 0) return _mod -P;
		else return 0;
	}
}

/***************************************/
/* ATOMIC ADDITION FUNCTION ON DOUBLES */
/***************************************/
__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    register unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

/*********************************/
/* ATOMIC MIN FUNCTION ON FLOATS */
/*********************************/
__device__ float atomicMin(float* address, float val)
{
	int* address_as_i = (int*)address;
	int old = *address_as_i, assumed;
	do {
		assumed = old;
		old = ::atomicCAS(address_as_i, assumed,
			__float_as_int(::fminf(val, __int_as_float(assumed))));
	} while (assumed != old);
	return __int_as_float(old);
}


